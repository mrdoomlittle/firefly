#include "hip/hip_runtime.h"
# include "draw.hpp"
__global__ void _draw_pixmap(int unsigned *__xoffset, int unsigned *__yoffset, boost::uint8_t *__pixels, boost::uint8_t *__pixmap, int unsigned *__xlen) {
	int unsigned curr_pixel = ((threadIdx.x + (*__xoffset)) + ((blockIdx.x + (*__yoffset)) * (*__xlen))) * 4;
	int unsigned pixmap_pos = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;

	__pixels[curr_pixel] = __pixmap[pixmap_pos];
	__pixels[curr_pixel + 1] = __pixmap[pixmap_pos + 1];
	__pixels[curr_pixel + 2] = __pixmap[pixmap_pos + 2];
	__pixels[curr_pixel + 3] = __pixmap[pixmap_pos + 3];
}

int unsigned *xoffset, *yoffset, *xlen;
boost::uint8_t *pixels, *pixmap;
void draw_pixmap(int unsigned __xoffset, int unsigned __yoffset, boost::uint8_t *__pixels, boost::uint8_t *__pixmap, int unsigned __sizes[2], int unsigned __imgsize[2], int unsigned __xlen) {
	hipMalloc((void **)&xoffset, sizeof(int unsigned));
	hipMalloc((void **)&yoffset, sizeof(int unsigned));
	hipMalloc((void **)&xlen, sizeof(int unsigned));	

	hipMalloc((void **)&pixels, __sizes[0]);
	hipMalloc((void **)&pixmap, __sizes[1]);

	hipMemcpy(xoffset, &__xoffset, sizeof(int unsigned), hipMemcpyHostToDevice);
	hipMemcpy(yoffset, &__yoffset, sizeof(int unsigned), hipMemcpyHostToDevice);
	hipMemcpy(xlen, &__xlen, sizeof(int unsigned), hipMemcpyHostToDevice);

	hipMemcpy(pixels, __pixels, __sizes[0], hipMemcpyHostToDevice);
	hipMemcpy(pixmap, __pixmap, __sizes[1], hipMemcpyHostToDevice);

	_draw_pixmap<<<__imgsize[1], __imgsize[0]>>>(xoffset, yoffset, pixels, pixmap, xlen);

	hipMemcpy(__pixels, pixels, __sizes[0], hipMemcpyDeviceToHost);

	hipFree(xoffset);
	hipFree(yoffset);
	hipFree(xlen);

	hipFree(pixels);
	hipFree(pixmap);
}
