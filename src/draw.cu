#include "hip/hip_runtime.h"
# include "draw.hpp"
__global__ void _draw_pixmap(int unsigned *__xoffset, int unsigned *__yoffset, boost::uint8_t *__pixels, boost::uint8_t *__pixmap, int unsigned *__xlen) {
	int unsigned curr_pixel = ((threadIdx.x + (*__xoffset)) + ((blockIdx.x + (*__yoffset)) * (*__xlen))) * 4;
	int unsigned pixmap_pos = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;

	__pixels[curr_pixel] = __pixmap[pixmap_pos];
	__pixels[curr_pixel + 1] = __pixmap[pixmap_pos + 1];
	__pixels[curr_pixel + 2] = __pixmap[pixmap_pos + 2];
	__pixels[curr_pixel + 3] = __pixmap[pixmap_pos + 3];
}

__global__ void _reset_pixmap(boost::uint8_t *__pixmap) {
	int unsigned pos = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;
	__pixmap[pos] = 0x0;
	__pixmap[pos + 1] = 0x0;
	__pixmap[pos + 2] = 0x0;
	__pixmap[pos + 3] = 0x0;
}

int unsigned *xoffset, *yoffset, *xlen;
boost::uint8_t *pixels, *pixmap;
void draw_pixmap(int unsigned __xoffset, int unsigned __yoffset, boost::uint8_t *__pixels, boost::uint8_t *__pixmap, int unsigned __sizes[2], int unsigned __imgsize[2], int unsigned __xlen) {
	hipMalloc((void **)&xoffset, sizeof(int unsigned));
	hipMalloc((void **)&yoffset, sizeof(int unsigned));
	hipMalloc((void **)&xlen, sizeof(int unsigned));	

	hipMalloc((void **)&pixels, __sizes[0]);
	hipMalloc((void **)&pixmap, __sizes[1]);

	hipMemcpy(xoffset, &__xoffset, sizeof(int unsigned), hipMemcpyHostToDevice);
	hipMemcpy(yoffset, &__yoffset, sizeof(int unsigned), hipMemcpyHostToDevice);
	hipMemcpy(xlen, &__xlen, sizeof(int unsigned), hipMemcpyHostToDevice);

	hipMemcpy(pixels, __pixels, __sizes[0], hipMemcpyHostToDevice);
	hipMemcpy(pixmap, __pixmap, __sizes[1], hipMemcpyHostToDevice);

	_draw_pixmap<<<__imgsize[1], __imgsize[0]>>>(xoffset, yoffset, pixels, pixmap, xlen);

	hipMemcpy(__pixels, pixels, __sizes[0], hipMemcpyDeviceToHost);

	hipFree(xoffset);
	hipFree(yoffset);
	hipFree(xlen);

	hipFree(pixels);
	hipFree(pixmap);
}

boost::uint8_t *bg_pixmap;
void reset_pixmap(boost::uint8_t *__pixmap, int unsigned __sizes[2]) {
	int unsigned size = (__sizes[0] * __sizes[1]) * 4;
	hipMalloc((void **)&bg_pixmap, size);

	hipMemcpy(bg_pixmap, __pixmap, size, hipMemcpyHostToDevice);

	_reset_pixmap<<<__sizes[1], __sizes[0]>>>(bg_pixmap);

	hipMemcpy(__pixmap, bg_pixmap, size, hipMemcpyDeviceToHost);

	hipFree(bg_pixmap);
}
