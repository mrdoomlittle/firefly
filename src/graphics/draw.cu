#include "hip/hip_runtime.h"
# include "draw.h"
# include "../system/io.h"
# include "../types/bool_t.h"
# include "../ffly_def.h"
# include "colour_attr.hpp"
# include "../system/errno.h"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../firefly.hpp"
__global__ void static pixdraw(mdl::uint_t __x, mdl::firefly::types::byte_t *__dst, mdl::uint_t __dstw, mdl::firefly::types::byte_t *__pixels) {
	mdl::firefly::types::byte_t *src = __pixels+((threadIdx.x+(blockIdx.x*blockDim.x))*4);
	mdl::firefly::types::byte_t *dst = __dst+((threadIdx.x+(blockIdx.x*__dstw)+__x)*4);

	mdl::firefly::types::byte_t alpha = *(src+A_OFF);
	mdl::firefly::types::byte_t invr_alpha = ~*(src+A_OFF);

	if (*(src+A_OFF) != 0x0) {
		mdl::firefly::types::byte_t r = (alpha**(src+R_OFF)+invr_alpha**(dst+R_OFF))>>8;
		mdl::firefly::types::byte_t g = (alpha**(src+G_OFF)+invr_alpha**(dst+G_OFF))>>8;
		mdl::firefly::types::byte_t b = (alpha**(src+B_OFF)+invr_alpha**(dst+B_OFF))>>8;
		*(dst+R_OFF) = r, *(dst+G_OFF) = g, *(dst+B_OFF) = b, *(dst+A_OFF) = *(src+A_OFF);
	}
}


mdl::firefly::types::byte_t static *dst = nullptr, *pixels = nullptr;
void static cleanup(void *__arg_p) {
	mdl::firefly::system::io::fprintf(ffly_log, "cleanup for draw.\n");
	if (dst != nullptr)
		mdl::firefly::memory::gpu_mem_free(dst);
	if (pixels != nullptr)
		mdl::firefly::memory::gpu_mem_free(pixels);
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_pixdraw(uint_t __x, uint_t __y, types::byte_t *__dst, uint_t __dstw, types::byte_t *__pixels, uint_t __width, uint_t __height) {
	types::cl_err_t err;
	types::bool_t static inited = ffly_false;
	uint_t dstsize = __dstw*__height*4;
	uint_t size = __width*__height*4;

	if (!inited) {
		if (memory::gpu_mem_alloc((void**)&dst, dstsize) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}

		if (memory::gpu_mem_alloc((void**)&pixels, size) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}

		ffly_act_add_task(&__ffly_act__, act_gid_cleanup, &cleanup, nullptr);
		inited = ffly_true;
	}

	uint_t static _dstsize = dstsize;
	if (_dstsize != dstsize) {
		if (dst != nullptr)
			mdl::firefly::memory::gpu_mem_free(dst);
		if (memory::gpu_mem_alloc((void**)&dst, dstsize) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}
		_dstsize = dstsize;
	}

	uint_t static _size = size;
	if (_size != size) {
		if (pixels != nullptr)
			mdl::firefly::memory::gpu_mem_free(pixels);
		if (memory::gpu_mem_alloc((void**)&pixels, size) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}
		_size = size;
	}

	if ((err = hipMemcpy(dst, __dst, dstsize, hipMemcpyHostToDevice)) != ffly_cl_success) {
		system::io::fprintf(ffly_err, "failed to copy dest to device, %s\n", hipGetErrorString(err));
		return FFLY_FAILURE;
	}

	if ((err = hipMemcpy(pixels, __pixels, size, hipMemcpyHostToDevice)) != ffly_cl_success) {
		system::io::fprintf(ffly_err, "failed to copy pixels to device, %s\n", hipGetErrorString(err));
		return FFLY_FAILURE;
	}

	// for now
	pixdraw<<<__height, __width>>>(__x, dst, __dstw, pixels);
	if ((err = hipMemcpy(__dst+((__y*__dstw)*4), dst, dstsize, hipMemcpyDeviceToHost)) != ffly_cl_success) {
		return FFLY_FAILURE;
	}
	return FFLY_SUCCESS;
}
