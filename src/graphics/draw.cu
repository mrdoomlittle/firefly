#include "hip/hip_runtime.h"
# include "draw.h"
# include "../system/io.h"
# include "../types/bool_t.h"
# include "../ffly_def.h"
# include "colour_attr.hpp"
# include "../system/errno.h"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../firefly.hpp"
__global__ void static pixdraw(mdl::firefly::types::off_t __xa, mdl::firefly::types::byte_t *__buff, mdl::uint_t __bufw, mdl::firefly::types::byte_t *__pixels) {
	mdl::firefly::types::byte_t *src = __pixels+((threadIdx.x+(blockIdx.x*blockDim.x))*4);
	mdl::firefly::types::byte_t *dst = __buff+((threadIdx.x+(blockIdx.x*__bufw)+__xa)*4);

	mdl::firefly::types::byte_t alpha = *(src+A_OFF);
	mdl::firefly::types::byte_t invr_alpha = ~*(src+A_OFF);

	if (*(src+A_OFF) != 0x0) {
		mdl::firefly::types::byte_t r = (alpha**(src+R_OFF)+invr_alpha**(dst+R_OFF))>>8;
		mdl::firefly::types::byte_t g = (alpha**(src+G_OFF)+invr_alpha**(dst+G_OFF))>>8;
		mdl::firefly::types::byte_t b = (alpha**(src+B_OFF)+invr_alpha**(dst+B_OFF))>>8;
		*(dst+R_OFF) = r, *(dst+G_OFF) = g, *(dst+B_OFF) = b, *(dst+A_OFF) = *(src+A_OFF);
	}
}


mdl::firefly::types::byte_t static *buff = nullptr, *pixels = nullptr;
void static cleanup(void *__arg_p) {
	mdl::firefly::system::io::fprintf(ffly_log, "cleanup for draw.\n");
	if (buff != nullptr)
		mdl::firefly::memory::gpu_mem_free(buff);
	if (pixels != nullptr)
		mdl::firefly::memory::gpu_mem_free(pixels);
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_pixdraw(types::off_t __xa, types::off_t __ya, types::byte_t *__buff, uint_t __bufw, types::byte_t *__pixels, uint_t __width, uint_t __height) {
	hipError_t any_err;
	types::bool_t static inited = ffly_false;
	uint_t bufsize = __bufw*__height*4;
	uint_t size = __width*__height*4;

	if (!inited) {
		if (memory::gpu_mem_alloc((void**)&buff, bufsize) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}

		if (memory::gpu_mem_alloc((void**)&pixels, size) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}

		ffly_act_add_task(&__ffly_act__, act_gid_cleanup, &cleanup, nullptr);
		inited = ffly_true;
	}

	uint_t static _bufsize = bufsize;
	if (_bufsize != bufsize) {
		if (buff != nullptr)
			mdl::firefly::memory::gpu_mem_free(buff);
		if (memory::gpu_mem_alloc((void**)&buff, bufsize) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}
		_bufsize = bufsize;
	}

	uint_t static _size = size;
	if (_size != size) {
		if (pixels != nullptr)
			mdl::firefly::memory::gpu_mem_free(pixels);
		if (memory::gpu_mem_alloc((void**)&pixels, size) != FFLY_SUCCESS) {
			return FFLY_FAILURE;
		}
		_size = size;
	}

	if ((any_err = hipMemcpy(buff, __buff, bufsize, hipMemcpyHostToDevice)) != hipSuccess) {
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy(pixels, __pixels, size, hipMemcpyHostToDevice)) != hipSuccess) {
		return FFLY_FAILURE;
	}

	// for now
	pixdraw<<<__height, __width>>>(__xa, buff, __bufw, pixels);
	if ((any_err = hipMemcpy(__buff+((__ya*__bufw)*4), buff, bufsize, hipMemcpyDeviceToHost)) != hipSuccess) {
		return FFLY_FAILURE;
	}
	return FFLY_SUCCESS;
}
