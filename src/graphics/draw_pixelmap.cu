#include "hip/hip_runtime.h"
# include "draw_pixelmap.hpp"
# include "../types/byte_t.h"
# include "colour_attr.hpp"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../ffly_def.h"
# include "../system/errno.h"
# include "../system/io.h"
__global__ void static draw_pixelmap(mdl::uint_t *__xfs, mdl::uint_t *__yfs, mdl::firefly::types::pixelmap_t __pixelbuff, mdl::uint_t *__pb_xa_len, mdl::uint_t *__pm_xfs, mdl::firefly::types::pixelmap_t __pixelmap, mdl::uint_t *__pm_xa_len, mdl::u16_t *__angle) {
	mdl::firefly::types::byte_t *src = __pixelmap+(((threadIdx.x+*__pm_xfs)+(blockIdx.x**__pm_xa_len))*4);
	mdl::firefly::types::byte_t *buf = __pixelbuff+(((threadIdx.x+*__xfs)+(blockIdx.x**__pb_xa_len))*4);

	mdl::firefly::types::byte_t alpha = *(src+A_OFF);
	mdl::firefly::types::byte_t invr_alpha = ~*(src+A_OFF);

	if (*(src+A_OFF) != 0x0) {
		mdl::firefly::types::byte_t r = (alpha**(src+R_OFF)+invr_alpha**(buf+R_OFF))>>8;
		mdl::firefly::types::byte_t g = (alpha**(src+G_OFF)+invr_alpha**(buf+G_OFF))>>8;
		mdl::firefly::types::byte_t b = (alpha**(src+B_OFF)+invr_alpha**(buf+B_OFF))>>8;
		*(buf+R_OFF) = r, *(buf+G_OFF) = g, *(buf+B_OFF) = b, *(buf+A_OFF) = *(src+A_OFF);
	}
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_draw_pixelmap(gpu_dr_pm_mem& __gpu_dr_pm_mem, uint_t __xfs, uint_t __yfs, types::pixelmap_t __pixelbuff, uint_t __pb_xa_len, uint_t __pb_ya_len, uint_t __pm_xfs, types::pixelmap_t __pixelmap, uint_t __pm_xa_len, uint_t __pm_ya_len, uint_t __rpm_xa_len, u16_t __angle) {
	types::err_t any_err;
	uint_t pixelbuff_size = __pm_ya_len*__pb_xa_len*4;
	uint_t pixelmap_size = __pm_ya_len*__rpm_xa_len*4;
	if (!__gpu_dr_pm_mem.inited) {
		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.xfs, sizeof(uint_t))) != FFLY_SUCCESS) {
			system::io::printf(stderr, "draw_pixelmap: cuda, failed to alloc memory for 'xfs', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.yfs, sizeof(uint_t))) != FFLY_SUCCESS) {
			system::io::printf(stderr, "draw_pixelmap: cuda, failed to alloc memory for 'yfs', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.pb_xa_len, sizeof(uint_t))) != FFLY_SUCCESS) {
			system::io::printf(stderr, "draw_pixelmap: cuda, failed to alloc memory for 'pb_xa_len', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.pm_xa_len, sizeof(uint_t))) != FFLY_SUCCESS) {
			system::io::printf(stderr, "draw_pixelmap: cuda, failed to alloc memory for 'pm_xa_len', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.pm_xfs, sizeof(uint_t))) != FFLY_SUCCESS) {
			system::io::printf(stderr, "draw_pixelmap: cuda, failed to alloc memory for 'pm_xfs', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.angle, sizeof(u16_t))) != FFLY_SUCCESS) {
			system::io::printf(stderr, "draw_pixelmap: cuda, failed to alloc memory for 'angle', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if (__xfs == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixelmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}

		if (__yfs == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixelmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}

		if (__pm_xfs == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixelmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}

		if (__angle == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.angle, &__angle, sizeof(u16_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixelmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}
		__gpu_dr_pm_mem.inited = ffly_true;
	}

	if (__gpu_dr_pm_mem._pb_size != pixelbuff_size)
	{
		if (__gpu_dr_pm_mem.pixelbuff != nullptr) hipFree(__gpu_dr_pm_mem.pixelbuff);
		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.pixelbuff, pixelbuff_size*sizeof(types::__pixelmap_t))) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Malloc, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pb_size = pixelbuff_size;
	}

	if (__gpu_dr_pm_mem._pm_size != pixelmap_size)
	{
		if (__gpu_dr_pm_mem.pixelmap != nullptr) hipFree(__gpu_dr_pm_mem.pixelmap);
		if ((any_err = memory::gpu_mem_alloc((void**)&__gpu_dr_pm_mem.pixelmap, pixelmap_size*sizeof(types::__pixelmap_t))) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Malloc, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pm_size = pixelmap_size;
	}

	if (__gpu_dr_pm_mem._xfs != __xfs)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._xfs = __xfs;
	}

	if (__gpu_dr_pm_mem._yfs != __yfs)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._yfs = __yfs;
	}

	if (__gpu_dr_pm_mem._pb_xa_len != __pb_xa_len)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pb_xa_len, &__pb_xa_len, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pb_xa_len = __pb_xa_len;
	}

	if (__gpu_dr_pm_mem._pm_xa_len != __rpm_xa_len)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pm_xa_len, &__rpm_xa_len, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pm_xa_len = __rpm_xa_len;
	}

	if (__gpu_dr_pm_mem._pm_xfs != __pm_xfs)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pm_xfs = __pm_xfs;
	}

	if (__gpu_dr_pm_mem._angle != __angle)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.angle, &__angle, sizeof(u16_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._angle = __angle;
	}

	uint_t pixelbuff_off = __yfs >= (__pb_ya_len-__pm_ya_len)? ((__pb_ya_len-__pm_ya_len)*__pb_xa_len)*4:(__yfs*__pb_xa_len)*4;
	if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pixelbuff, __pixelbuff+pixelbuff_off, pixelbuff_size*sizeof(types::__pixelmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTCM;
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pixelmap, __pixelmap, pixelmap_size*sizeof(types::__pixelmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTCM;
		return FFLY_FAILURE;
	}

	draw_pixelmap<<<__pm_ya_len, __pm_xa_len>>>(__gpu_dr_pm_mem.xfs, __gpu_dr_pm_mem.yfs, __gpu_dr_pm_mem.pixelbuff, __gpu_dr_pm_mem.pb_xa_len, __gpu_dr_pm_mem.pm_xfs, __gpu_dr_pm_mem.pixelmap, __gpu_dr_pm_mem.pm_xa_len, __gpu_dr_pm_mem.angle);
	if ((any_err = hipDeviceSynchronize()) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to sync device, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTSD;
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy(__pixelbuff+pixelbuff_off, __gpu_dr_pm_mem.pixelbuff, pixelbuff_size*sizeof(types::__pixelmap_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTCM;
		return FFLY_FAILURE;
	}
	return FFLY_SUCCESS;
}
