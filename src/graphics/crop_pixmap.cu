#include "hip/hip_runtime.h"
# include "crop_pixmap.hpp"
__global__ void cu_crop_pixmap(mdl::uint_t *__xfs, mdl::firefly::types::pixmap_t __pixbuff, mdl::firefly::types::pixmap_t __pixmap, mdl::uint_t *__pm_xlen, mdl::u8_t *__chn_c) {
	mdl::uint_t pb_point = (threadIdx.x + (blockIdx.x * blockDim.x)) * *__chn_c;
	mdl::uint_t pm_point = (threadIdx.x + *__xfs + (blockIdx.x * *__pm_xlen)) * *__chn_c;
	for (mdl::u8_t chn{}; chn != *__chn_c; chn ++) __pixbuff[pb_point + chn] = __pixmap[pm_point + chn];
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_crop_pixmap(uint_t __xfs, uint_t __yfs, types::pixmap_t __pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, types::pixmap_t __pixmap, uint_t __pm_xlen, uint_t __pm_ylen, u8_t __chn_c) {
	bool static inited = false;
	static types::pixmap_t pixbuff = nullptr, pixmap = nullptr;
	static uint_t *xfs = nullptr, *pm_xlen;
	static u8_t *chn_c = nullptr;

	hipError_t any_err = hipSuccess;
	uint_t pb_size = __pb_xlen * __pb_ylen * __chn_c;
	uint_t pm_size = __pb_ylen * __pm_xlen * __chn_c;
	if (!inited) {
		if ((any_err = hipMalloc((void **)&xfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'xfs', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&pm_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'pb_xlen', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&chn_c, sizeof(u8_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'chn_c', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if (__xfs == 0) {
			if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'xfs' to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}

		if (__chn_c == 0) {
			if ((any_err = hipMemcpy(chn_c, &__chn_c, sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'chn_c' to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}
		inited = true;
	}

	uint_t static _pb_size{};
	if (_pb_size != pb_size)
	{
		if (pixbuff != nullptr) hipFree(pixbuff);
		if ((any_err = hipMalloc((void **)&pixbuff, pb_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'pixbuff', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pb_size = pb_size;
	}

	uint_t static _pm_size{};
	if (_pm_size != pm_size)
	{
		if (pixmap != nullptr) hipFree(pixmap);
		if ((any_err = hipMalloc((void **)&pixmap, pm_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'pixmap', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_size = pm_size;
	}

	uint_t static _xfs{};
	if (_xfs != __xfs) {
		if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_xfs = __xfs;
	}

	uint_t static _pm_xlen{};
	if (_pm_xlen != __pm_xlen) {
		if ((any_err = hipMemcpy(pm_xlen, &__pm_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pb_xlen' to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_xlen = __pm_xlen;
	}

	u8_t static _chn_c{};
	if (_chn_c != __chn_c) {
		if ((any_err = hipMemcpy(chn_c, &__chn_c, sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'chn_c' to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_chn_c = __chn_c;
	}

	uint_t pm_fs = __yfs >= (__pm_ylen - __pb_ylen)? ((__pm_ylen - __pb_ylen) * __pm_xlen) * __chn_c : __yfs * __pm_xlen * __chn_c;
	if ((any_err = hipMemcpy(pixbuff, __pixbuff, pb_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixbuff' to device, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy(pixmap, __pixmap + pm_fs, pm_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixmap' to device, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	cu_crop_pixmap<<<__pb_ylen, __pb_xlen>>>(xfs, pixbuff, pixmap, pm_xlen, chn_c);
	if ((any_err = hipMemcpy(__pixbuff, pixbuff, pb_size * sizeof(types::__pixmap_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixmap' to host, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	return FFLY_SUCCESS;
}
