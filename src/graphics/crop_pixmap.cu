#include "hip/hip_runtime.h"
# include "crop_pixmap.hpp"
__global__ void cu_crop_pixmap(mdl::uint_t *__xfs, mdl::firefly::types::pixmap_t __pixbuff, mdl::firefly::types::pixmap_t __pixmap, mdl::uint_t *__pm_xlen, mdl::u8_t *__chn_c) {
	mdl::uint_t pb_point = (threadIdx.x + (blockIdx.x * blockDim.x)) * *__chn_c;
	mdl::uint_t pm_point = (threadIdx.x + *__xfs + (blockIdx.x * *__pm_xlen)) * *__chn_c;
	for (mdl::u8_t chn{}; chn != *__chn_c; chn ++) __pixbuff[pb_point + chn] = __pixmap[pm_point + chn];
}

__global__ void cu_crop_2d_pm(mdl::uint_t *__xfs, mdl::firefly::types::_1d_pm_t __pixbuff, mdl::firefly::types::_1d_pm_t __pixmap, mdl::u8_t *__chn_c, std::size_t __pb_pitch, std::size_t __pm_pitch) {
	mdl::firefly::types::_1d_pm_t pb_row = __pixbuff + blockIdx.x * __pb_pitch;
	mdl::firefly::types::_1d_pm_t pm_row = __pixmap + blockIdx.x * __pm_pitch;


//	for (int x = 0; x != 64; x ++) pb_row[x] = 'H';
	//printf("%d\n", threadIdx.x);
	//printf("%d - %d - %d\n", *__xfs, *__chn_c, (mdl::i8_t)pm_row[0]);
	for (mdl::u8_t chn{}; chn != *__chn_c; chn ++) pb_row[(threadIdx.x * *__chn_c) + chn] = pm_row[((threadIdx.x + *__xfs) * *__chn_c) + chn];
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_crop_2d_pm(uint_t __xfs, uint_t __yfs, types::_2d_pm_t __pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, uint_t __pb_rxlen,
	types::_2d_pm_t __pixmap, uint_t __pm_xlen, uint_t __pm_ylen, uint_t __pm_rxlen, u8_t __chn_c) {
	bool static inited = false;
	types::_1d_pm_t static pixbuff = nullptr, pixmap = nullptr;
	uint_t static *xfs = nullptr;
	u8_t static *chn_c = nullptr;

	hipError_t any_err = hipSuccess;
	uint_t pb_size = __pb_xlen * __pb_ylen * __chn_c;
	uint_t pm_size = __pm_xlen * __pm_ylen * __chn_c;
	if (!inited) {
		if ((any_err = hipMalloc((void **)&xfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'xfs', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&chn_c, sizeof(u8_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'chn_c', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if (__xfs == 0) {
			if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'xfs' to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}

		if (__chn_c == 0) {
			if ((any_err = hipMemcpy(chn_c, &__chn_c, sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'chn_c' to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}
		inited = true;
	}

	std::size_t static pb_pitch{};
	uint_t static _pb_size{};
	if (_pb_size != pb_size) {
		if (pixbuff != nullptr) hipFree(pixbuff);
		if ((any_err = hipMallocPitch((void **)&pixbuff, &pb_pitch, __pb_xlen * __chn_c * sizeof(types::__pixmap_t), __pb_ylen)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for pixbuff, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pb_size = pb_size;
	}

	std::size_t static pm_pitch{};
	uint_t static _pm_size{};

	if (_pm_size != pm_size) {
		if (pixmap != nullptr) hipFree(pixmap);
		if ((any_err = hipMallocPitch((void**)&pixmap, &pm_pitch, __pm_xlen * __chn_c, __pm_ylen - __yfs)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for pixmap, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_size = pm_size;
	}

	uint_t static _xfs{};
	if (_xfs != __xfs) {
		if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'xfs' to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_xfs = __xfs;
	}

	u8_t static _chn_c{};
	if (_chn_c != __chn_c) {
		if ((any_err = hipMemcpy(chn_c, &__chn_c, sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'chn_c' to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_chn_c = __chn_c;
	}

	if ((any_err = hipMemcpy2D(pixbuff, pb_pitch, __pixbuff[0], __pb_rxlen * __chn_c, __pb_xlen * __chn_c, __pb_ylen, hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixbuff' to device, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy2D(pixmap, pm_pitch, __pixmap[__yfs], __pm_rxlen * __chn_c, __pm_xlen * __chn_c, __pb_ylen - __yfs, hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixmap' to device, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	printf("%d - %d\n", __pb_xlen, __pb_ylen);
	cu_crop_2d_pm<<<__pb_ylen, __pb_xlen>>>(xfs, pixbuff, pixmap, chn_c, pb_pitch, pm_pitch);
	hipDeviceSynchronize();
/*
	if ((any_err = hipMemcpy2D(__pixmap + __yfs, __pm_xlen * __chn_c, pixmap, pm_pitch, __pm_xlen * __chn_c, __pb_ylen, hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixmap' to host, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}
*/
	if ((any_err = hipMemcpy2D(__pixbuff[0], __pb_rxlen * __chn_c, pixbuff, pb_pitch, __pb_xlen * __chn_c, __pb_ylen, hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixmap' to host, errno: %d\n", any_err);
	}

	return FFLY_SUCCESS;
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_crop_pixmap(uint_t __xfs, uint_t __yfs, types::pixmap_t __pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, types::pixmap_t __pixmap, uint_t __pm_xlen, uint_t __pm_ylen, u8_t __chn_c) {
	bool static inited = false;
	types::pixmap_t static pixbuff = nullptr, pixmap = nullptr;
	uint_t static *xfs = nullptr, *pm_xlen;
	u8_t static *chn_c = nullptr;

	hipError_t any_err = hipSuccess;
	uint_t pb_size = __pb_xlen * __pb_ylen * __chn_c;
	uint_t pm_size = __pb_ylen * __pm_xlen * __chn_c;
	if (!inited) {
		if ((any_err = hipMalloc((void **)&xfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'xfs', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&pm_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'pb_xlen', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&chn_c, sizeof(u8_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'chn_c', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if (__xfs == 0) {
			if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'xfs' to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}

		if (__chn_c == 0) {
			if ((any_err = hipMemcpy(chn_c, &__chn_c, sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'chn_c' to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}
		inited = true;
	}

	uint_t static _pb_size{};
	if (_pb_size != pb_size)
	{
		if (pixbuff != nullptr) hipFree(pixbuff);
		if ((any_err = hipMalloc((void **)&pixbuff, pb_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'pixbuff', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pb_size = pb_size;
	}

	uint_t static _pm_size{};
	if (_pm_size != pm_size)
	{
		if (pixmap != nullptr) hipFree(pixmap);
		if ((any_err = hipMalloc((void **)&pixmap, pm_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to alloc memory for 'pixmap', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_size = pm_size;
	}

	uint_t static _xfs{};
	if (_xfs != __xfs) {
		if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_xfs = __xfs;
	}

	uint_t static _pm_xlen{};
	if (_pm_xlen != __pm_xlen) {
		if ((any_err = hipMemcpy(pm_xlen, &__pm_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pb_xlen' to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_xlen = __pm_xlen;
	}

	u8_t static _chn_c{};
	if (_chn_c != __chn_c) {
		if ((any_err = hipMemcpy(chn_c, &__chn_c, sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'chn_c' to device, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_chn_c = __chn_c;
	}

	if ((any_err = hipMemcpy(pixbuff, __pixbuff, pb_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixbuff' to device, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}
	uint_t pm_fs = __yfs >= (__pm_ylen - __pb_ylen)? ((__pm_ylen - __pb_ylen) * __pm_xlen) * __chn_c : __yfs * __pm_xlen * __chn_c;

	if ((any_err = hipMemcpy(pixmap, __pixmap + pm_fs, pm_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixmap' to device, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	cu_crop_pixmap<<<__pb_ylen, __pb_xlen>>>(xfs, pixbuff, pixmap, pm_xlen, chn_c);
	if ((any_err = hipMemcpy(__pixbuff, pixbuff, pb_size * sizeof(types::__pixmap_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "crop_pixmap: cuda, failed to copy memory for 'pixmap' to host, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	return FFLY_SUCCESS;
}
