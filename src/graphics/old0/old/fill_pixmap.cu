#include "hip/hip_runtime.h"
# include "fill_pixmap.hpp"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
__global__ void cu_fill_pixmap(mdl::firefly::types::pixmap_t __pixmap, mdl::firefly::types::colour_t *__colour) {
	mdl::uint_t point = (threadIdx.x+(blockIdx.x*blockDim.x))*4;

	__pixmap[point] =__colour-> r;
	__pixmap[point+1] =__colour-> g;
	__pixmap[point+2] =__colour-> b;
	__pixmap[point+3] =__colour-> a;
}


mdl::firefly::types::err_t mdl::firefly::graphics::fill_pixmap(types::pixmap_t __pixmap, uint_t __xa_len, uint_t __ya_len, types::colour_t __colour) {
	types::pixmap_t static pixmap = nullptr;
	types::colour_t static *colour = nullptr;
	bool static inited = false;
	hipError_t any_error = hipSuccess;

	uint_t pixmap_size = __xa_len*__ya_len*4;
	uint_t static _pixmap_size = 0;

	if (pixmap_size == 0) {
		system::io::printf(stderr, "cuda, fill_pixmap: error pixmap size must not be zero.\n");
		return FFLY_FAILURE;
	}

	if (_pixmap_size != pixmap_size) {
		if (pixmap != nullptr) memory::gpu_mem_free(pixmap);

		if (memory::gpu_mem_alloc((void**)&pixmap, pixmap_size*sizeof(types::__pixmap_t)) != FFLY_SUCCESS) {
			system::io::printf(stderr, "cuda, fill_pixmap: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_pixmap_size = pixmap_size;
	}

	if (!inited) {
		if (memory::gpu_mem_alloc((void**)&colour, sizeof(types::colour_t)) != FFLY_SUCCESS) {
			system::io::printf(stderr, "cuda, fill_pixmap: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if (__colour.r == 0 && __colour.g == 0 && __colour.b == 0 && __colour.a == 0) {
			if ((any_error = hipMemcpy(colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "cuda, fill_pixmap: failed to call Memcpy, error code: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		inited = true;
	}

	hipMemcpy(pixmap, __pixmap, pixmap_size*sizeof(types::__pixmap_t), hipMemcpyHostToDevice);

	types::colour_t static _colour = {0, 0, 0, 0};
	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		if ((any_error = hipMemcpy(colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda, fill_pixmap: failed to call Memcpy, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_colour = __colour;
	}

	cu_fill_pixmap<<<__ya_len, __xa_len>>>(pixmap, colour);
	hipMemcpy(__pixmap, pixmap, pixmap_size*sizeof(u8_t), hipMemcpyDeviceToHost);
	return FFLY_SUCCESS;
}
