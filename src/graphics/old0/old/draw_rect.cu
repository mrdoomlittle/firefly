#include "hip/hip_runtime.h"
# include "draw_rect.hpp"
__global__ void cu_draw_rect(boost::uint8_t *__pixbuff, mdl::uint_t *__b_xcoord, mdl::uint_t *__b_ycoord, mdl::firefly::graphics::colour_t *__colour, mdl::uint_t *__pd_xlen) {
	mdl::uint_t point = ((threadIdx.x + (*__b_xcoord)) + ((blockIdx.x + (*__b_ycoord)) * (*__pd_xlen))) * 4;

	__pixbuff[point] = __colour-> r;
	__pixbuff[point + 1] = __colour-> g;
	__pixbuff[point + 2] = __colour-> b;
	__pixbuff[point + 3] = __colour-> a;
}


boost::uint8_t *pixbuff;
mdl::uint_t *b_xcoord, *b_ycoord, *pd_xlen;
mdl::firefly::graphics::colour_t *colour;
boost::int8_t mdl::firefly::graphics::draw_rect(boost::uint8_t *__pixbuff, uint_t __b_xcoord, uint_t __b_ycoord, uint_t __e_xcoord, uint_t __e_ycoord, colour_t __colour, uint_t __pd_xlen, uint_t __pd_ylen) {
	uint_t pixbuff_size = (__pd_xlen * __pd_ylen) * 4;

	hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(boost::uint8_t));
	hipMalloc((void **)&b_xcoord, sizeof(uint_t));
	hipMalloc((void **)&b_ycoord, sizeof(uint_t));
	hipMalloc((void **)&colour, sizeof(colour_t));
	hipMalloc((void **)&pd_xlen, sizeof(uint_t));

	hipMemcpy(pixbuff, __pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(b_xcoord, &__b_xcoord, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(b_ycoord, &__b_ycoord, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice);
	hipMemcpy(pd_xlen, &__pd_xlen, sizeof(uint_t), hipMemcpyHostToDevice);

	cu_draw_rect<<<(__e_ycoord - __b_ycoord), (__e_xcoord - __b_xcoord)>>>(
		pixbuff,
		b_xcoord,
		b_ycoord,
		colour,
		pd_xlen
	);

	hipMemcpy(__pixbuff, pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost);

	hipFree(pixbuff);
	hipFree(b_xcoord);
	hipFree(b_ycoord);
	hipFree(colour);
	hipFree(pd_xlen);
}
