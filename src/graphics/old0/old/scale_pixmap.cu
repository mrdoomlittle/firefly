#include "hip/hip_runtime.h"
# include "scale_pixmap.hpp"

__global__ void cu_scale_pixmap(boost::uint8_t *__pixmap, boost::uint8_t *__pixbuff, mdl::uint_t *__scale) {
	// dont know if this way is better then copying the memory over every time
	// may change.
	mdl::uint_t pixbuff_xlen = (gridDim.x * (*__scale));
	mdl::uint_t pixmap_point = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;
	mdl::uint_t pixbuff_point = ((threadIdx.x + (blockIdx.x * pixbuff_xlen)) * 4) * (*__scale);

	for (std::size_t y = 0; y != (*__scale); y ++) {
		for (std::size_t x = 0; x != (*__scale); x ++) {
			mdl::uint_t real_point = pixbuff_point + ((x + (y * pixbuff_xlen)) * 4);

			__pixbuff[real_point] = __pixmap[pixmap_point];
			__pixbuff[real_point + 1] = __pixmap[pixmap_point + 1];
			__pixbuff[real_point + 2] = __pixmap[pixmap_point + 2];
			__pixbuff[real_point + 3] = __pixmap[pixmap_point + 3];
		}
	}
}

boost::int8_t mdl::firefly::graphics::scale_pixmap(boost::uint8_t *&__pixmap, uint_t& __pixmap_xlen, uint_t& __pixmap_ylen, uint_t __scale) {
	if (__scale == 0) {
		fprintf(stderr, "error: scale must be above 0.\n");
		return -1;
	}

	uint_t pixmap_size = (__pixmap_xlen * __pixmap_ylen) * 4;
	static uint_t _pixmap_size = 0;

	static boost::uint8_t *pixmap = nullptr, *pixbuff = nullptr;
	static uint_t new_size = 0;
	boost::uint8_t *_pixbuff = nullptr;
	hipError_t any_error = hipSuccess;

	if (_pixmap_size != pixmap_size) {
		if (pixmap != nullptr) hipFree(pixmap);

		if ((any_error = hipMalloc((void **)&pixmap, pixmap_size * sizeof(boost::uint8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n");
			return -1;
		}

		uint_t new_xlen = __pixmap_xlen * __scale;
		uint_t new_ylen = __pixmap_ylen * __scale;
		new_size = (new_xlen * new_ylen) * 4;

		if (pixbuff != nullptr) hipFree(pixbuff);

		if ((any_error = hipMalloc((void **)&pixbuff, new_size * sizeof(boost::uint8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n");
			return -1;
		}

		_pixbuff = static_cast<boost::uint8_t *>(malloc(new_size * sizeof(boost::uint8_t)));
	}

	static uint_t *scale = nullptr;
	static bool initialized = false;

	if (!initialized) {
		if ((any_error = hipMalloc((void **)&scale, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		initialized = true;
	}

	static uint_t _scale = 0;

	if (_scale != __scale) {
		if ((any_error = hipMemcpy(scale, &__scale, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		}

		_scale = __scale;
	}


	if ((any_error = hipMemcpy(pixmap, __pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		return -1;
	}

	cu_scale_pixmap<<<__pixmap_ylen, __pixmap_xlen>>>(pixmap, pixbuff, scale);

	if ((any_error = hipMemcpy(_pixbuff, pixbuff, new_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n");
		return -1;
	}

	std::free(__pixmap);
	__pixmap = _pixbuff;
	__pixmap_xlen = __pixmap_xlen * __scale;
	__pixmap_ylen = __pixmap_ylen * __scale;

	return 0;
}
