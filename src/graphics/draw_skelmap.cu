#include "hip/hip_runtime.h"
# include "draw_skelmap.hpp"
__global__ void cu_draw_skelmap(mdl::u8_t *__skelmap, mdl::u8_t *__pixbuff, mdl::uint_t *__xaxis, mdl::uint_t *__yaxis, mdl::uint_t *__pixbuff_xlen, mdl::uint_t *__scale, mdl::firefly::graphics::colour_t *__colour) {
	mdl::uint_t skel_point = threadIdx.x + (blockIdx.x * blockDim.x);

if (__skelmap[skel_point] == 0x1) {
	mdl::uint_t pix_point = (((threadIdx.x + (*__xaxis)) + ((blockIdx.x + (*__yaxis)) * (*__pixbuff_xlen))) * 4) * (*__scale);

	for (std::size_t y = 0; y != (*__scale); y ++) {
		for (std::size_t x = 0; x != (*__scale); x ++) {
			mdl::uint_t real_pp = pix_point + ((x + (y * (*__pixbuff_xlen))) * 4);
			__pixbuff[real_pp] = __colour-> r;
			__pixbuff[real_pp + 1] = __colour-> g;
			__pixbuff[real_pp + 2] = __colour-> b;
			__pixbuff[real_pp + 3] = __colour-> a;
		}
	}
}
}

mdl::i8_t mdl::firefly::graphics::draw_skelmap(u8_t *__skelmap, u8_t *__pixbuff, uint_t __xaxis, uint_t __yaxis, types::skelmap_info_t __skelmap_info, uint_t __pixbuff_xlen, uint_t __pixbuff_ylen, uint_t __scale, colour_t __colour) {
	static u8_t *skelmap = nullptr, *pixbuff = nullptr;
	static uint_t *xaxis = nullptr, *yaxis = nullptr, *pixbuff_xlen = nullptr, *scale = nullptr;
	static colour_t *colour = nullptr;
	static bool initialized = false;
	hipError_t any_error = hipSuccess;

	uint_t pixbuff_size = (__pixbuff_xlen * __pixbuff_ylen) * 4;

	if (pixbuff_size == 0) {
		fprintf(stderr, "error: pixbuff size is zero.\n");
		return -1;
	}

	if (!initialized) {
		if ((any_error = hipMalloc((void **)&xaxis, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		if (__xaxis == 0) {
			if ((any_error = hipMemcpy(xaxis, &__xaxis, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return -1;	
			}
		}

		if ((any_error = hipMalloc((void **)&yaxis, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		if (__yaxis == 0) {
			if ((any_error = hipMemcpy(yaxis, &__yaxis, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return -1;
			}
		}

		if ((any_error = hipMalloc((void **)&pixbuff_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		if ((any_error = hipMalloc((void **)&scale, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		if (__scale == 0) {
			if ((any_error = hipMemcpy(scale, &__scale, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return -1;
			}
		}

		if ((any_error = hipMalloc((void **)&colour, sizeof(colour_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		if (__colour.r == 0 && __colour.g == 0 && __colour.b == 0 && __colour.a == 0) {
			if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return -1;
			}
		}

		initialized = true;
	}

	static types::skelmap_info_t _skelmap_info = {0, 0};

	if (_skelmap_info.xaxis_len != __skelmap_info.xaxis_len || _skelmap_info.yaxis_len != __skelmap_info.yaxis_len) {
		if (skelmap != nullptr) hipFree(skelmap);

		if ((any_error = hipMalloc((void **)&skelmap, (__skelmap_info.xaxis_len * __skelmap_info.yaxis_len) * sizeof(u8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		_skelmap_info = __skelmap_info;
	}

	static uint_t _pixbuff_size = 0;
	static uint_t _pixbuff_xlen = 0;

	if (_pixbuff_size != pixbuff_size) {
		if (pixbuff != nullptr) hipFree(pixbuff);

		if ((any_error = hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(u8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		if (_pixbuff_xlen != __pixbuff_xlen) {
			if ((any_error = hipMemcpy(pixbuff_xlen, &__pixbuff_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return -1;
			}

			_pixbuff_xlen = __pixbuff_xlen;
		}

		_pixbuff_size = pixbuff_size;
	}

	if ((any_error = hipMemcpy(skelmap, __skelmap, (__skelmap_info.xaxis_len * __skelmap_info.yaxis_len) * sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		return -1;
	}

	if ((any_error = hipMemcpy(pixbuff, __pixbuff, pixbuff_size * sizeof(u8_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		return -1;
	}

	static uint_t _xaxis = 0;
	if (_xaxis != __xaxis) {
		if ((any_error = hipMemcpy(xaxis, &__xaxis, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return -1;
		}

		_xaxis = __xaxis;
	}

	static uint_t _yaxis = 0;
	if (_yaxis != __yaxis) {
		if ((any_error = hipMemcpy(yaxis, &__yaxis, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return -1;
		}

		_yaxis = __yaxis;
	}

	static uint_t _scale = 0;
	if (_scale != __scale) {
		if ((any_error = hipMemcpy(scale, &__scale, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return -1;
		}

		_scale = __scale;
	}

	static colour_t _colour;

	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return -1;
		}

		_colour = __colour;
	}

	cu_draw_skelmap<<<__skelmap_info.yaxis_len, __skelmap_info.xaxis_len>>>(skelmap, pixbuff, xaxis, yaxis, pixbuff_xlen, scale, colour);

	if ((any_error = hipMemcpy(__pixbuff, pixbuff, pixbuff_size * sizeof(u8_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		return -1;
	}

	return 0;
}
