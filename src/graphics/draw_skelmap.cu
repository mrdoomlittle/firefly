#include "hip/hip_runtime.h"
# include "draw_skelmap.hpp"
__global__ void cu_draw_skelmap(boost::uint8_t *__skelmap, boost::uint8_t *__pixbuff, mdl::uint_t *__xaxis, mdl::uint_t *__yaxis, mdl::uint_t *__pixbuff_xlen, mdl::uint_t *__scale, mdl::firefly::graphics::colour_t *__colour) {
	mdl::uint_t skel_point = threadIdx.x + (blockIdx.x * blockDim.x);

if (__skelmap[skel_point] == 0x1) {
	mdl::uint_t pix_point = (((threadIdx.x + (*__xaxis)) + ((blockIdx.x + (*__yaxis)) * (*__pixbuff_xlen))) * 4) * (*__scale);

	for (std::size_t y = 0; y != (*__scale); y ++) {
		for (std::size_t x = 0; x != (*__scale); x ++) {
			mdl::uint_t real_pp = pix_point + ((x + (y * (*__pixbuff_xlen))) * 4);
			__pixbuff[real_pp] = __colour-> r;
			__pixbuff[real_pp + 1] = __colour-> g;
			__pixbuff[real_pp + 2] = __colour-> b;
			__pixbuff[real_pp + 3] = __colour-> a;
		}
	}
}
}

void mdl::firefly::graphics::draw_skelmap(boost::uint8_t *__skelmap, boost::uint8_t *__pixbuff, uint_t __xaxis, uint_t __yaxis, types::skelmap_info_t __skelmap_info, uint_t __pixbuff_xlen, uint_t __pixbuff_ylen, uint_t __scale, colour_t __colour) {
	static boost::uint8_t *skelmap = nullptr, *pixbuff = nullptr;
	static uint_t *xaxis = nullptr, *yaxis = nullptr, *pixbuff_xlen = nullptr, *scale = nullptr;
	static colour_t *colour = nullptr;
	static bool initialized = false;

	uint_t pixbuff_size = (__pixbuff_xlen * __pixbuff_ylen) * 4;

	if (!initialized) {
		hipMalloc((void **)&xaxis, sizeof(uint_t));
		hipMalloc((void **)&yaxis, sizeof(uint_t));
		hipMalloc((void **)&pixbuff_xlen, sizeof(uint_t));
		hipMalloc((void **)&scale, sizeof(uint_t));
		hipMalloc((void **)&colour, sizeof(colour_t));

		initialized = true;
	}

	static types::skelmap_info_t _skelmap_info = {0, 0};

	if (_skelmap_info.xaxis_len != __skelmap_info.xaxis_len || _skelmap_info.yaxis_len != __skelmap_info.yaxis_len) {
		if (skelmap != nullptr) hipFree(skelmap);

		hipMalloc((void **)&skelmap, (__skelmap_info.xaxis_len * __skelmap_info.yaxis_len) * sizeof(boost::uint8_t));

		_skelmap_info = __skelmap_info;
	}

	static uint_t _pixbuff_size = 0;
	static uint_t _pixbuff_xlen = 0;

	if (_pixbuff_size != pixbuff_size) {
		if (pixbuff != nullptr) hipFree(pixbuff);

		hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(boost::uint8_t));

		if (_pixbuff_xlen != __pixbuff_xlen) {
			hipMemcpy(pixbuff_xlen, &__pixbuff_xlen, sizeof(uint_t), hipMemcpyHostToDevice);
			_pixbuff_xlen = __pixbuff_xlen;
		}

		_pixbuff_size = pixbuff_size;
	}

	hipMemcpy(skelmap, __skelmap, (__skelmap_info.xaxis_len * __skelmap_info.yaxis_len) * sizeof(boost::uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(pixbuff, __pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice);

	static uint_t _xaxis = 0;
	if (_xaxis != __xaxis) {
		hipMemcpy(xaxis, &__xaxis, sizeof(uint_t), hipMemcpyHostToDevice);
		_xaxis = __xaxis;
	}

	static uint_t _yaxis = 0;
	if (_yaxis != __yaxis) {
		hipMemcpy(yaxis, &__yaxis, sizeof(uint_t), hipMemcpyHostToDevice);
		_yaxis = __yaxis;
	}

	static uint_t _scale = 0;
	if (_scale != __scale) {
		hipMemcpy(scale, &__scale, sizeof(uint_t), hipMemcpyHostToDevice);
		_scale = __scale;
	}

	static colour_t _colour;

	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice);
		_colour = __colour;
	}

	cu_draw_skelmap<<<__skelmap_info.yaxis_len, __skelmap_info.xaxis_len>>>(skelmap, pixbuff, xaxis, yaxis, pixbuff_xlen, scale, colour);

	hipMemcpy(__pixbuff, pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost);
}
