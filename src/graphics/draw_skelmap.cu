#include "hip/hip_runtime.h"
# include "draw_skelmap.hpp"
__global__ void cu_draw_skelmap(boost::uint8_t *__skelmap, boost::uint8_t *__pixbuff, mdl::uint_t *__xaxis, mdl::uint_t *__yaxis, mdl::uint_t *__pixbuff_xlen, mdl::uint_t *__scale, mdl::firefly::graphics::colour_t *__colour) {
	mdl::uint_t skel_point = threadIdx.x + (blockIdx.x * blockDim.x);
	if (__skelmap[skel_point] == 0x0) return;

	mdl::uint_t pix_point = (((threadIdx.x + (*__xaxis)) + ((blockIdx.x + (*__yaxis)) * (*__pixbuff_xlen))) * 4) * (*__scale);

	for (std::size_t y = 0; y != (*__scale); y ++) {
		for (std::size_t x = 0; x != (*__scale); x ++) {
			mdl::uint_t real_pp = pix_point + ((x + (y * (*__pixbuff_xlen))) * 4);
			__pixbuff[real_pp] = __colour-> r;
			__pixbuff[real_pp + 1] = __colour-> g;
			__pixbuff[real_pp + 2] = __colour-> b;
			__pixbuff[real_pp + 3] = __colour-> a;
		}
	}
}

void mdl::firefly::graphics::draw_skelmap(boost::uint8_t *__skelmap, boost::uint8_t *__pixbuff, uint_t __xaxis, uint_t __yaxis, types::skelmap_info_t __skelmap_info, uint_t __pixbuff_xlen, uint_t __pixbuff_ylen, uint_t __scale, colour_t __colour) {
	boost::uint8_t *skelmap, *pixbuff;
	uint_t *xaxis, *yaxis, *pixbuff_xlen, *scale;
	colour_t *colour;

	uint_t pixbuff_size = (__pixbuff_xlen * __pixbuff_ylen) * 4;
	hipMalloc((void **)&skelmap, (__skelmap_info.xaxis_len * __skelmap_info.yaxis_len) * sizeof(boost::uint8_t));
	hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(boost::uint8_t));
	hipMalloc((void **)&xaxis, sizeof(uint_t));
	hipMalloc((void **)&yaxis, sizeof(uint_t));
	hipMalloc((void **)&pixbuff_xlen, sizeof(uint_t));
	hipMalloc((void **)&scale, sizeof(uint_t));
	hipMalloc((void **)&colour, sizeof(colour_t));

	hipMemcpy(skelmap, __skelmap, (__skelmap_info.xaxis_len * __skelmap_info.yaxis_len) * sizeof(boost::uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(pixbuff, __pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(xaxis, &__xaxis, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(yaxis, &__yaxis, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(pixbuff_xlen, &__pixbuff_xlen, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(scale, &__scale, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice);

	cu_draw_skelmap<<<__skelmap_info.yaxis_len, __skelmap_info.xaxis_len>>>(skelmap, pixbuff, xaxis, yaxis, pixbuff_xlen, scale, colour);

	hipMemcpy(__pixbuff, pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost);

	hipFree(skelmap);
	hipFree(pixbuff);	
	hipFree(xaxis);
	hipFree(yaxis);
	hipFree(pixbuff_xlen);
	hipFree(scale);
	hipFree(colour);
}
