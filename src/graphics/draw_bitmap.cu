#include "hip/hip_runtime.h"
# include "draw_bitmap.hpp"
__global__ void cu_draw_bitmap(mdl::firefly::types::bitmap_t __bitmap, mdl::firefly::types::pixmap_t __pixbuff, mdl::uint_t *__pb_xlen, mdl::firefly::types::coords_t *__coords, mdl::firefly::graphics::colour_t *__colour) {
	mdl::uint_t bit_point = threadIdx.x + (blockIdx.x * blockDim.x);
	mdl::uint_t pix_point = ((threadIdx.x + __coords-> xaxis) + ((blockIdx.x + __coords-> yaxis) * (*__pb_xlen))) * 4;

	__pixbuff[pix_point] = __colour-> r * __bitmap[bit_point] / 255;
	__pixbuff[pix_point + 1] = __colour-> g * __bitmap[bit_point] / 255;
	__pixbuff[pix_point + 2] = __colour-> b * __bitmap[bit_point] / 255;
	__pixbuff[pix_point + 3] = __colour-> a * __bitmap[bit_point] / 255;
}

boost::int8_t mdl::firefly::graphics::draw_bitmap(types::bitmap_t __bitmap, uint_t __bm_xlen, uint_t __bm_ylen, types::pixmap_t __pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, types::coords_t __coords, colour_t __colour) {
	static types::bitmap_t bitmap = nullptr;
	static types::pixmap_t pixbuff = nullptr;
	static uint_t *pb_xlen = nullptr;
	static types::coords_t *coords = nullptr;
	static colour_t *colour = nullptr;
	static bool initialized = false;
	hipError_t any_error = hipSuccess;

	if (!initialized) {
		if ((any_error = hipMalloc((void **)&pb_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if ((any_error = hipMalloc((void **)&coords, sizeof(types::coords_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}


		if ((any_error = hipMalloc((void **)&colour, sizeof(colour_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}


		if (__pb_xlen == 0) {
			if ((any_error = hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}


		if (__coords.xaxis == 0 && __coords.yaxis == 0) {
			if ((any_error = hipMemcpy(coords, &__coords, sizeof(types::coords_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__colour.r == 0 && __colour.g == 0 && __colour.b == 0 && __colour.a == 0) {
			if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		initialized = true;
	}

	uint_t bitmap_size = __bm_xlen * __bm_ylen;
	uint_t pixbuff_size = (__pb_xlen * __pb_ylen) * 4;

	static uint_t _pb_xlen = 0;
	if (_pb_xlen != __pb_xlen) {
		if ((any_error = hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_pb_xlen = __pb_xlen;
	}

	static types::coords_t _coords = {
		.xaxis = 0,
		.yaxis = 0
	};
	if (_coords.xaxis != __coords.xaxis || _coords.yaxis != __coords.yaxis) {
		if ((any_error = hipMemcpy(coords, &__coords, sizeof(types::coords_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_coords = __coords;
	}


	static colour_t _colour = {
		.r = 0,
		.g = 0,
		.b = 0,
		.a = 0
	};
	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_colour = __colour;
	}

	static uint_t _bitmap_size = 0;
	if (_bitmap_size != bitmap_size) {
		if (bitmap != nullptr) {
			if ((any_error = hipFree(bitmap)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call hipFree, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}
		if ((any_error = hipMalloc((void **)&bitmap, bitmap_size * sizeof(boost::uint8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_bitmap_size = bitmap_size;
	}

	static uint_t _pixbuff_size = 0;
	if (_pixbuff_size != pixbuff_size) {
		if (pixbuff != nullptr) { 
			if ((any_error = hipFree(pixbuff)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call hipFree, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}
		if ((any_error = hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(boost::uint8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_pixbuff_size = pixbuff_size;
	}

	if ((any_error = hipMemcpy(bitmap, __bitmap, bitmap_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
		return FFLY_FAILURE;
	}

	if ((any_error = hipMemcpy(pixbuff, __pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
		return FFLY_FAILURE;
	}

	cu_draw_bitmap<<<__bm_ylen, __bm_xlen>>>(bitmap, pixbuff, pb_xlen, coords, colour);

	if ((any_error = hipMemcpy(__pixbuff, pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
		return FFLY_FAILURE;
	}	

	return FFLY_SUCCESS;
}
