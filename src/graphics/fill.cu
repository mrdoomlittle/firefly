#include "hip/hip_runtime.h"
# include "fill.h"
# include "../system/io.h"
# include "../types/bool_t.h"
# include "../ffly_def.h"
# include "colour_attr.hpp"
# include "../system/errno.h"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../firefly.hpp"
# include "../system/thread.h"
__global__ void static pixfill(mdl::firefly::types::byte_t *__dst, mdl::firefly::types::colour_t *__colour) {
	mdl::firefly::types::byte_t *p = __dst+((threadIdx.x+(blockIdx.x*blockDim.x))*4);
	*(p+R_OFF) = __colour->r, *(p+G_OFF) = __colour->g, *(p+B_OFF) = __colour->b, *(p+A_OFF) = __colour->a;
}

// mutithreading hasen't been tested on this and may not work.

struct context {
    mdl::firefly::types::byte_t *dev_dst;
    mdl::firefly::types::colour_t *dev_colour;
    mdl::firefly::types::colour_t colour;
    mdl_uint_t size;
    mdl::firefly::types::bool_t prepared;
};

/*
    how many threads can access at one time.
*/
# define THREADS 8
static struct context parent_ctx = {
    dev_dst:nullptr,
    dev_colour:nullptr,
    colour:{r:0, g:0, b:0, a:0},
    size:0,
    prepared:ffly_false
};

static struct context ctx_list[THREADS];
void static cleanup(void *__arg_p) {
	mdl::firefly::system::io::fprintf(ffly_log, "cleanup for fill.\n");
    mdl_uint_t i = 0;
    while(i != THREADS) {
        struct context *ctx = &ctx_list[i];
    	if (ctx->dev_dst != nullptr)
    		mdl::firefly::memory::gpu_mem_free(ctx->dev_dst);
    	if (ctx->dev_colour != nullptr)
    		mdl::firefly::memory::gpu_mem_free(ctx->dev_colour);
        i++;
    }
}
# include "../system/err.h"
# include "../system/mutex.h"
ffly_mutex_t static mutex = FFLY_MUTEX_INIT;
mdl::firefly::types::err_t mdl::firefly::graphics::gpu_pixfill(types::byte_t *__dst, mdl_uint_t __nopix, types::colour_t __colour) {
	types::cl_err_t err;
	mdl_uint_t size = __nopix*4;

    ffly_tid_t tid = ffly_gettid();
    // thread id should only be null if its the parent prossess calling
    struct context *ctx = tid == FFLY_TID_NULL?&parent_ctx:&ctx_list[tid];
    types::bool_t static inited = ffly_false;
    if (!inited) {
        if (_ok(ffly_mutex_trylock(&mutex))) {
            if (!inited) {
                mdl_uint_t i = 0;
                while(i != THREADS) {
                    ctx[i++] = (struct context) {
                        dev_dst:nullptr,
                        dev_colour:nullptr,
                        colour:{r:0, g:0, b:0, a:0},
                        size:0,
                        prepared:ffly_false
                    };
                }
                ffly_act_add_task(&__ffly_act__, act_gid_cleanup, &cleanup, nullptr);
                inited = ffly_true;
            }
            ffly_mutex_unlock(&mutex);
        }
    }
 
    while(!inited);
    if (!ctx->prepared) {
		if (memory::gpu_mem_alloc((void**)&ctx->dev_colour, sizeof(types::colour_t)) != FFLY_SUCCESS) {
			system::io::fprintf(ffly_err, "cuda, failed to allocate memory for colour.\n");
			return FFLY_FAILURE;
		}

		if (memory::gpu_mem_alloc((void**)&ctx->dev_dst, size) != FFLY_SUCCESS) {
			system::io::fprintf(ffly_err, "cuda, failed to allocate memory for dest.\n");
			return FFLY_FAILURE;
		}

		if ((err = hipMemcpy(ctx->dev_colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != ffly_cl_success) {
			system::io::fprintf(ffly_err, "cuda, failed to copy colour to device, %s\n", hipGetErrorString(err));
			return FFLY_FAILURE;
		}
		ctx->prepared = ffly_true;
	}

	if (ctx->size != size) {
		if (ctx->dev_dst != nullptr) memory::gpu_mem_free(ctx->dev_dst);
		if (memory::gpu_mem_alloc((void**)&ctx->dev_dst, size) != FFLY_SUCCESS) {
			system::io::fprintf(ffly_err, "cuda, failed to allocate memory for dest.\n");
			return FFLY_FAILURE;
		}
		ctx->size = size;
	}

	if ((err = hipMemcpy(ctx->dev_dst, __dst, size, hipMemcpyHostToDevice)) != ffly_cl_success) {
		system::io::fprintf(ffly_err, "cuda, failed to copy dest to device, %s\n", hipGetErrorString(err));
		return FFLY_FAILURE;
	}

	if (ctx->colour.r != __colour.r || ctx->colour.g != __colour.g || ctx->colour.b != __colour.b || ctx->colour.a != __colour.a) {
		if ((err = hipMemcpy(ctx->dev_colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != ffly_cl_success) {
			system::io::fprintf(ffly_err, "cuda, failed to copy colour to device.\n");
			return FFLY_FAILURE;
		}
		ctx->colour = __colour;
	}

	mdl_uint_t blk_size;
	mdl_uint_t no_blks, left, off;

	if (!(__nopix>>8)) {
		blk_size = __nopix;
		no_blks = 1;
	} else {
		blk_size = 1<<8;
		no_blks = __nopix>>8;
	}

	system::io::fprintf(ffly_log, "no_blks: %u, blk_size: %u\n", no_blks, blk_size);
	pixfill<<<no_blks, blk_size>>>(ctx->dev_dst, ctx->dev_colour);
	if ((left = (__nopix-(off = (no_blks*(1<<8)))))>0 && (__nopix>>8)>0)
		pixfill<<<1, left>>>(ctx->dev_dst+(off*4), ctx->dev_colour);
	if ((err = hipMemcpy(__dst, ctx->dev_dst, size, hipMemcpyDeviceToHost)) != ffly_cl_success) {
		system::io::fprintf(ffly_err, "cuda, failed to copy dest from device to host.\n");
		return FFLY_FAILURE;
	}
	return FFLY_SUCCESS;
}
