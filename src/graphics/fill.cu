#include "hip/hip_runtime.h"
# include "fill.h"
# include "../system/io.h"
# include "../types/bool_t.h"
# include "../ffly_def.h"
# include "colour_attr.hpp"
# include "../system/errno.h"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../firefly.hpp"
__global__ void static pixfill(mdl::firefly::types::byte_t *__buff, mdl::firefly::types::colour_t *__colour) {
	mdl::firefly::types::byte_t *p = __buff+((threadIdx.x+(blockIdx.x*blockDim.x))*4);
	*(p+R_OFF) = __colour->r, *(p+G_OFF) = __colour->g, *(p+B_OFF) = __colour->b, *(p+A_OFF) = __colour->a;
}

mdl::firefly::types::byte_t static *buff = nullptr;
mdl::firefly::types::colour_t static *colour = nullptr;
void static cleanup(void *__arg_p) {
	mdl::firefly::system::io::fprintf(ffly_log, "cleanup for fill.\n");
	if (buff != nullptr)
		mdl::firefly::memory::gpu_mem_free(buff);
	if (colour != nullptr)
		mdl::firefly::memory::gpu_mem_free(colour);
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_pixfill(types::byte_t *__buff, mdl_uint_t __nopix, types::colour_t __colour) {
	hipError_t any_err;
	types::bool_t static inited = ffly_false;
	mdl_uint_t size = __nopix*4;
	if (!inited) {
		if (memory::gpu_mem_alloc((void**)&colour, sizeof(types::colour_t)) != FFLY_SUCCESS) {
			system::io::fprintf(ffly_err, "cuda, failed to allocate memory for colour.\n");
			return FFLY_FAILURE;
		}

		if (memory::gpu_mem_alloc((void**)&buff, size) != FFLY_SUCCESS) {
			system::io::fprintf(ffly_err, "cuda, failed to allocate memory for buff.\n");
			return FFLY_FAILURE;
		}

		if ((any_err = hipMemcpy(buff, __buff, size, hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::fprintf(ffly_err, "cuda, failed to copy buff to device.\n");
			return FFLY_FAILURE;
		}

		if ((any_err = hipMemcpy(colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::fprintf(ffly_err, "cuda, failed to copy colour to device.\n");
			return FFLY_FAILURE;
		}
		ffly_act_add_task(&__ffly_act__, act_gid_cleanup, &cleanup, nullptr);
		inited = ffly_true;
	}

	mdl_uint_t static _size = size;
	if (_size != size) {
		if (buff != nullptr) memory::gpu_mem_free(buff);
		if (memory::gpu_mem_alloc((void**)&buff, size) != FFLY_SUCCESS) {
			system::io::fprintf(ffly_err, "cuda, failed to allocate memory for buff.\n");
			return FFLY_FAILURE;
		}
		_size = size;
	}


	if ((any_err = hipMemcpy(buff, __buff, size, hipMemcpyHostToDevice)) != hipSuccess) {
		system::io::fprintf(ffly_err, "cuda, failed to copy buff to device.\n");
		return FFLY_FAILURE;
	}

	types::colour_t static _colour = __colour;
	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		if ((any_err = hipMemcpy(colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::fprintf(ffly_err, "cuda, failed to copy colour to device.\n");
			return FFLY_FAILURE;
		}
		_colour = __colour;
	}

	mdl_uint_t blk_size;
	mdl_uint_t no_blks, left, off;

	if (!(__nopix>>8)) {
		blk_size = __nopix;
		no_blks = 1;
	} else {
		blk_size = 1<<8;
		no_blks = __nopix>>8;
	}

	system::io::fprintf(ffly_log, "no_blks: %u, blk_size: %u\n", no_blks, blk_size);
	pixfill<<<no_blks, blk_size>>>(buff, colour);
	if ((left = (__nopix-(off = (no_blks*(1<<8)))))>0 && (__nopix>>8)>0)
		pixfill<<<1, left>>>(buff+(off*4), colour);
	if ((any_err = hipMemcpy(__buff, buff, size, hipMemcpyDeviceToHost)) != hipSuccess) {
		system::io::fprintf(ffly_err, "cuda, failed to copy buff from device to host.\n");
		return FFLY_FAILURE;
	}
	return FFLY_SUCCESS;
}
