#include "hip/hip_runtime.h"
# include "fill.h"
# include "../system/io.h"
# include "../types/bool_t.h"
# include "../ffly_def.h"
# include "colour_attr.hpp"
# include "../system/errno.h"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../firefly.hpp"
__global__ void static fill_frame(mdl::firefly::types::byte_t *__frame, mdl::firefly::types::colour_t *__colour) {
	mdl::firefly::types::byte_t *p = __frame+((threadIdx.x+(blockIdx.x*blockDim.x))*4);
	*(p+R_OFF) = __colour->r, *(p+G_OFF) = __colour->g, *(p+B_OFF) = __colour->b, *(p+A_OFF) = __colour->a;
}

mdl::firefly::types::byte_t static *frame;
mdl::firefly::types::colour_t static *colour;
void static cleanup(void *__arg_p) {
	mdl::firefly::memory::gpu_mem_free(frame);
	mdl::firefly::memory::gpu_mem_free(colour);
}

mdl::firefly::types::err_t mdl::firefly::graphics::fill_frame(types::byte_t *__frame, mdl_uint_t __width, mdl_uint_t __height, types::colour_t __colour) {
	hipError_t any_err;
	types::bool_t inited = ffly_false;
	mdl_uint_t size = __width*__height*4;
	if (!inited) {
		if (memory::gpu_mem_alloc((void**)&colour, sizeof(types::colour_t)) != FFLY_SUCCESS) {
			system::io::printf(ffly_err, "failed to allocate memory for colour.\n");
			return FFLY_FAILURE;
		}

		if (memory::gpu_mem_alloc((void**)&frame, size*sizeof(types::byte_t)) != FFLY_SUCCESS) {
			system::io::printf(ffly_err, "failed to allocate memory for frame.\n");
			return FFLY_FAILURE;
		}

		hipMemcpy(frame, __frame, size*sizeof(types::byte_t), hipMemcpyHostToDevice);
		if ((any_err = hipMemcpy(colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(ffly_err, "failed to copy colour to device.\n");
			return FFLY_FAILURE;
		}

		ffly_act_add_task(&__ffly_act__, act_gid_cleanup, &cleanup, NULL);
		inited = ffly_true;
	}

	mdl_uint_t static _size = size;
	if (_size != size) {
		if (frame != nullptr) memory::gpu_mem_free(frame);
		if (memory::gpu_mem_alloc((void**)&frame, size*sizeof(types::byte_t)) != FFLY_SUCCESS) {
			system::io::printf(ffly_err, "failed to allocate memory for frame.\n");
			return FFLY_FAILURE;
		}
		_size = size;
	}

	types::colour_t static _colour = __colour;
	hipMemcpy(frame, __frame, size*sizeof(types::byte_t), hipMemcpyHostToDevice);
	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		if ((any_err = hipMemcpy(colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(ffly_err, "failed to copy colour to device.\n");
			return FFLY_FAILURE;
		}
		_colour = __colour;
	}

	fill_frame<<<__height, __width>>>(frame, colour);
	hipMemcpy(__frame, frame, size*sizeof(types::byte_t), hipMemcpyDeviceToHost);
	return FFLY_SUCCESS;
}
