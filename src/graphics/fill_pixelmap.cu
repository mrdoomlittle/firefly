#include "hip/hip_runtime.h"
# include "fill_pixelmap.hpp"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../system/errno.h"
# include "../ffly_def.h"
# include "../system/io.h"
# include "../types/byte_t.h"
# include "colour_attr.hpp"
__global__ void static fill_pixelmap(mdl::firefly::types::pixelmap_t __pixelmap, mdl::firefly::types::colour_t *__colour) {
	mdl::firefly::types::byte_t *p = __pixelmap+((threadIdx.x+(blockIdx.x*blockDim.x))*4);
	*(p+R_OFF) = __colour->r, *(p+G_OFF) = __colour->g, *(p+B_OFF) = __colour->b, *(p+A_OFF) = __colour->a;
}

void mdl::firefly::graphics::gpu_fill_pixelmap(gpu_fl_pm_mem& __mem, types::pixelmap_t __pixelmap, mdl_uint_t __xa_len, mdl_uint_t __ya_len, types::colour_t __colour) {
	mdl_uint_t pixelmap_size = __xa_len*__ya_len*4;
	hipError_t any_err;

	if (!__mem.inited) {
		printf("init.\n");
		__mem.pixelmap_size = pixelmap_size;
		if (memory::gpu_mem_alloc((void**)&__mem.colour, sizeof(types::colour_t)) != FFLY_SUCCESS) {
			system::io::printf(stderr, "cuda failed.\n");
			return;
		}		

		if (memory::gpu_mem_alloc((void**)&__mem.pixelmap, pixelmap_size*sizeof(types::__pixelmap_t)) != FFLY_SUCCESS) {
			system::io::printf(stderr, "cuda failed.\n");
			return;
		}

		__mem.inited = ffly_true;
	}

	if (__mem.pixelmap_size != pixelmap_size) {
		if (__mem.pixelmap != nullptr) memory::gpu_mem_free(__mem.pixelmap);
		if (memory::gpu_mem_alloc((void**)&__mem.pixelmap, pixelmap_size*sizeof(types::__pixelmap_t)) != FFLY_SUCCESS) {
			system::io::printf(stderr, "cuda failed.\n");
			return;
		}
		__mem.pixelmap_size = pixelmap_size;
	}

	hipMemcpy(__mem.pixelmap, __pixelmap, pixelmap_size*sizeof(types::__pixelmap_t), hipMemcpyHostToDevice);
	if (__mem._colour.r != __colour.r || __mem._colour.g != __colour.g || __mem._colour.b != __colour.b || __mem._colour.a != __colour.a) {
		if ((any_err = hipMemcpy(__mem.colour, &__colour, sizeof(types::colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda failed.\n");
			return;
		}
		__mem._colour = __colour;
	}

	fill_pixelmap<<<__ya_len, __xa_len>>>(__mem.pixelmap, __mem.colour);
	hipMemcpy(__pixelmap, __mem.pixelmap, pixelmap_size*sizeof(types::__pixelmap_t), hipMemcpyDeviceToHost);
}
