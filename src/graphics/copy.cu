# include "copy.h"
# include "../system/io.h"
# include "../types/bool_t.h"
# include "../ffly_def.h"
# include "colour_attr.hpp"
# include "../system/errno.h"
# include "../memory/mem_alloc.h"
# include "../memory/mem_free.h"
# include "../firefly.hpp"
mdl::firefly::types::err_t mdl::firefly::graphics::gpu_pixcopy(types::byte_t *__dst, types::byte_t *__src, mdl_uint_t __nopix) {
	hipError_t any_err;
	mdl_uint_t size = __nopix*4;
	hipMemcpy(__dst, __src, size, hipMemcpyHostToHost);
	hipDeviceSynchronize();
	return FFLY_SUCCESS;
}
