#include "hip/hip_runtime.h"
# include "fill_pixmap.hpp"
__global__ void cu_fill_pixmap(boost::uint8_t *__pixmap, mdl::firefly::graphics::colour_t *__colour) {
	mdl::uint_t point = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;

	__pixmap[point] =__colour-> r;
	__pixmap[point + 1] =__colour-> g;
	__pixmap[point + 2] =__colour-> b;
	__pixmap[point + 3] =__colour-> a;
}


void mdl::firefly::graphics::fill_pixmap(boost::uint8_t *__pixmap, uint_t __xlen, uint_t __ylen, colour_t __colour) {
	boost::uint8_t *pixmap;
	colour_t *colour;
	uint_t pixmap_size = (__xlen * __ylen) * 4;

	hipMalloc((void **)&pixmap, pixmap_size * sizeof(boost::uint8_t));
	hipMalloc((void **)&colour, sizeof(colour_t));


	hipMemcpy(pixmap, __pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice);

	cu_fill_pixmap<<<__ylen, __xlen>>>(pixmap, colour);

	hipMemcpy(__pixmap, pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost);

	hipFree(pixmap);
	hipFree(colour);
}
