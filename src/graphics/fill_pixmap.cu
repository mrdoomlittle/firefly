#include "hip/hip_runtime.h"
# include "fill_pixmap.hpp"
__global__ void cu_fill_pixmap(boost::uint8_t *__pixmap, mdl::firefly::graphics::colour_t *__colour) {
	mdl::uint_t point = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;

	__pixmap[point] =__colour-> r;
	__pixmap[point + 1] =__colour-> g;
	__pixmap[point + 2] =__colour-> b;
	__pixmap[point + 3] =__colour-> a;
}


boost::int8_t mdl::firefly::graphics::fill_pixmap(boost::uint8_t *__pixmap, uint_t __xlen, uint_t __ylen, colour_t __colour) {
	static boost::uint8_t *pixmap = nullptr;
	static colour_t *colour = nullptr;
	static bool initialized = false;
	hipError_t any_error = hipSuccess;

	uint_t pixmap_size = (__xlen * __ylen) * 4;
	static uint_t _pixmap_size = 0;

	if (pixmap_size == 0) {
		fprintf(stderr, "error pixmap size must not be zero.\n");
		return -1;
	}

	if (_pixmap_size != pixmap_size) {
		if (pixmap == nullptr) hipFree(pixmap);

		if ((any_error = hipMalloc((void **)&pixmap, pixmap_size * sizeof(boost::uint8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		} 
	}

	if (!initialized) {
		if ((any_error = hipMalloc((void **)&colour, sizeof(colour_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return -1;
		}

		if (__colour.r == 0 && __colour.g == 0 && __colour.b == 0 && __colour.a == 0) {
			if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return -1;
			}
		}
	}

	hipMemcpy(pixmap, __pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice);

	static colour_t _colour = {0, 0, 0, 0};

	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return -1;
		}
	}

	cu_fill_pixmap<<<__ylen, __xlen>>>(pixmap, colour);

	hipMemcpy(__pixmap, pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost);

	return 0;
}
