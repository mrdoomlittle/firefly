#include "hip/hip_runtime.h"
# include "fill_pixmap.hpp"
__global__ void cu_fill_pixmap(mdl::firefly::types::pixmap_t __pixmap, mdl::firefly::graphics::colour_t *__colour) {
	mdl::uint_t point = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;

	__pixmap[point] =__colour-> r;
	__pixmap[point + 1] =__colour-> g;
	__pixmap[point + 2] =__colour-> b;
	__pixmap[point + 3] =__colour-> a;
}


mdl::firefly::types::err_t mdl::firefly::graphics::fill_pixmap(types::pixmap_t __pixmap, uint_t __xaxis_len, uint_t __yaxis_len, colour_t __colour) {
	static types::pixmap_t pixmap = nullptr;
	static colour_t *colour = nullptr;
	static bool initialized = false;
	hipError_t any_error = hipSuccess;

	uint_t pixmap_size = (__xaxis_len * __yaxis_len) * 4;
	static uint_t _pixmap_size = 0;

	if (pixmap_size == 0) {
		fprintf(stderr, "error pixmap size must not be zero.\n");
		return FFLY_FAILURE;
	}

	if (_pixmap_size != pixmap_size) {
		if (pixmap != nullptr) hipFree(pixmap);

		if ((any_error = hipMalloc((void **)&pixmap, pixmap_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		} 
	}

	if (!initialized) {
		if ((any_error = hipMalloc((void **)&colour, sizeof(colour_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if (__colour.r == 0 && __colour.g == 0 && __colour.b == 0 && __colour.a == 0) {
			if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		initialized = true;
	}

	hipMemcpy(pixmap, __pixmap, pixmap_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice);

	static colour_t _colour = {0, 0, 0, 0};

	if (_colour.r != __colour.r || _colour.g != __colour.g || _colour.b != __colour.b || _colour.a != __colour.a) {
		if ((any_error = hipMemcpy(colour, &__colour, sizeof(colour_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}
	}

	cu_fill_pixmap<<<__yaxis_len, __xaxis_len>>>(pixmap, colour);

	hipMemcpy(__pixmap, pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost);

	return FFLY_SUCCESS;
}
