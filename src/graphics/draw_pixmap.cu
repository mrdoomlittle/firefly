#include "hip/hip_runtime.h"
# include "draw_pixmap.hpp"
__global__ void cu_draw_pixmap(mdl::uint_t *__xfs, mdl::uint_t *__yfs, mdl::firefly::types::pixmap_t __pixbuff, mdl::uint_t *__pb_xlen, mdl::uint_t *__pm_xfs, mdl::firefly::types::pixmap_t __pixmap, mdl::uint_t *__pm_xlen, mdl::u16_t *__angle) {
	mdl::uint_t pb_point = ((threadIdx.x+*__xfs)+(blockIdx.x**__pb_xlen))*4;
	mdl::uint_t pm_point = ((threadIdx.x+*__pm_xfs)+(blockIdx.x**__pm_xlen))*4;

	mdl::firefly::types::byte_t alpha = __pixmap[pm_point+3];
	mdl::firefly::types::byte_t inv_alpha = 255-__pixmap[pm_point+3];

	if (__pixmap[pm_point+3] != 0x0) {
		mdl::firefly::types::byte_t r = (mdl::firefly::types::byte_t)((alpha*__pixmap[pm_point]+inv_alpha*__pixbuff[pb_point]) >> 8);
		mdl::firefly::types::byte_t g = (mdl::firefly::types::byte_t)((alpha*__pixmap[pm_point+1]+inv_alpha*__pixbuff[pb_point+1]) >> 8);
		mdl::firefly::types::byte_t b = (mdl::firefly::types::byte_t)((alpha*__pixmap[pm_point+2]+inv_alpha*__pixbuff[pb_point+2]) >> 8);

		__pixbuff[pb_point] = r;
		__pixbuff[pb_point+1] = g;
		__pixbuff[pb_point+2] = b;
		__pixbuff[pb_point+3] = __pixmap[pm_point+3];
	}
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_draw_pixmap(gpu_dr_pm_mem& __gpu_dr_pm_mem, uint_t __xfs, uint_t __yfs, types::pixmap_t __pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, uint_t __pm_xfs, types::pixmap_t __pixmap, uint_t __pm_xlen, uint_t __pm_ylen, uint_t __rpm_xlen, u16_t __angle) {
	types::cu_err_t any_err = hipSuccess;
	uint_t pb_size = __pm_ylen*__pb_xlen*4;
	uint_t pm_size = __pm_ylen*__rpm_xlen*4;
	if (!__gpu_dr_pm_mem.inited) {
		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.xfs, sizeof(uint_t))) != hipSuccess) {
			system::io::printf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'xfs', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.yfs, sizeof(uint_t))) != hipSuccess) {
			system::io::printf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'yfs', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.pb_xlen, sizeof(uint_t))) != hipSuccess) {
			system::io::printf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'pb_xlen', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.pm_xlen, sizeof(uint_t))) != hipSuccess) {
			system::io::printf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'pm_xlen', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.pm_xfs, sizeof(uint_t))) != hipSuccess) {
			system::io::printf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'pm_xfs', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.angle, sizeof(u16_t))) != hipSuccess) {
			system::io::printf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'angle', cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}

		if (__xfs == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}

		if (__yfs == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}

		if (__pm_xfs == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}

		if (__angle == 0) {
			if ((any_err = hipMemcpy(__gpu_dr_pm_mem.angle, &__angle, sizeof(u16_t), hipMemcpyHostToDevice)) != hipSuccess) {
				system::io::printf(stderr, "draw_pixmap: cuda, failed to copy memory to device, cu_errno: %d\n", any_err);
				ffly_errno = FF_ERR_CFTCM;
				return FFLY_FAILURE;
			}
		}
		__gpu_dr_pm_mem.inited = true;
	}

	if (__gpu_dr_pm_mem._pb_size != pb_size)
	{
		if (__gpu_dr_pm_mem.pixbuff != nullptr) hipFree(__gpu_dr_pm_mem.pixbuff);
		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.pixbuff, pb_size*sizeof(types::__pixmap_t))) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Malloc, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pb_size = pb_size;
	}

	if (__gpu_dr_pm_mem._pm_size != pm_size)
	{
		if (__gpu_dr_pm_mem.pixmap != nullptr) hipFree(__gpu_dr_pm_mem.pixmap);
		if ((any_err = hipMalloc((void**)&__gpu_dr_pm_mem.pixmap, pm_size*sizeof(types::__pixmap_t))) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Malloc, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTAM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pm_size = pm_size;
	}

	if (__gpu_dr_pm_mem._xfs != __xfs)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._xfs = __xfs;
	}

	if (__gpu_dr_pm_mem._yfs != __yfs)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._yfs = __yfs;
	}

	if (__gpu_dr_pm_mem._pb_xlen != __pb_xlen)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pb_xlen = __pb_xlen;
	}

	if (__gpu_dr_pm_mem._pm_xlen != __rpm_xlen)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pm_xlen, &__rpm_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pm_xlen = __rpm_xlen;
	}

	if (__gpu_dr_pm_mem._pm_xfs != __pm_xfs)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._pm_xfs = __pm_xfs;
	}

	if (__gpu_dr_pm_mem._angle != __angle)
	{
		if ((any_err = hipMemcpy(__gpu_dr_pm_mem.angle, &__angle, sizeof(u16_t), hipMemcpyHostToDevice)) != hipSuccess) {
			system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
			ffly_errno = FF_ERR_CFTCM;
			return FFLY_FAILURE;
		}
		__gpu_dr_pm_mem._angle = __angle;
	}

	uint_t pb_fs = __yfs >= (__pb_ylen-__pm_ylen)? ((__pb_ylen-__pm_ylen)*__pb_xlen)*4 : (__yfs*__pb_xlen)*4;
	if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pixbuff, __pixbuff+pb_fs, pb_size*sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTCM;
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy(__gpu_dr_pm_mem.pixmap, __pixmap, pm_size*sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTCM;
		return FFLY_FAILURE;
	}

	cu_draw_pixmap<<<__pm_ylen, __pm_xlen>>>(__gpu_dr_pm_mem.xfs, __gpu_dr_pm_mem.yfs, __gpu_dr_pm_mem.pixbuff, __gpu_dr_pm_mem.pb_xlen,
		__gpu_dr_pm_mem.pm_xfs, __gpu_dr_pm_mem.pixmap, __gpu_dr_pm_mem.pm_xlen, __gpu_dr_pm_mem.angle);
	if ((any_err = hipDeviceSynchronize()) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to sync device, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTSD;
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy(__pixbuff+pb_fs, __gpu_dr_pm_mem.pixbuff, pb_size*sizeof(types::__pixmap_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		system::io::printf(stderr, "cuda: failed to call Memcpy, cu_errno: %d\n", any_err);
		ffly_errno = FF_ERR_CFTCM;
		return FFLY_FAILURE;
	}
	return FFLY_SUCCESS;
}
