#include "hip/hip_runtime.h"
# include "draw_pixmap.hpp"
__global__ void cu_draw_pixmap(mdl::uint_t *__xfs, mdl::uint_t *__yfs, mdl::firefly::types::pixmap_t __pixbuff, mdl::uint_t *__pb_xlen, mdl::uint_t *__pm_xfs, mdl::firefly::types::pixmap_t __pixmap, mdl::uint_t *__pm_xlen, mdl::u16_t *__angle) {
	mdl::uint_t pb_point = ((threadIdx.x + *__xfs) + (blockIdx.x * *__pb_xlen)) * 4;
	mdl::uint_t pm_point = ((threadIdx.x + *__pm_xfs) + (blockIdx.x * *__pm_xlen)) * 4;

	mdl::firefly::types::byte_t alpha = __pixmap[pm_point + 3];
	mdl::firefly::types::byte_t inv_alpha = 255 - __pixmap[pm_point + 3];

	if (__pixmap[pm_point + 3] != 0x0) {
		mdl::firefly::types::byte_t new_r = (mdl::firefly::types::byte_t)((alpha * __pixmap[pm_point] + inv_alpha * __pixbuff[pb_point]) >> 8);
		mdl::firefly::types::byte_t new_g = (mdl::firefly::types::byte_t)((alpha * __pixmap[pm_point + 1] + inv_alpha * __pixbuff[pb_point + 1]) >> 8);
		mdl::firefly::types::byte_t new_b = (mdl::firefly::types::byte_t)((alpha * __pixmap[pm_point + 2] + inv_alpha * __pixbuff[pb_point + 2]) >> 8);

		__pixbuff[pb_point] = new_r;
		__pixbuff[pb_point + 1] = new_g;
		__pixbuff[pb_point + 2] = new_b;

		__pixbuff[pb_point + 3] = __pixmap[pm_point + 3];
	}
}

mdl::firefly::types::err_t mdl::firefly::graphics::gpu_draw_pixmap(uint_t __xfs, uint_t __yfs, types::pixmap_t __pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, uint_t __pm_xfs, types::pixmap_t __pixmap, uint_t __pm_xlen, uint_t __pm_ylen, uint_t __rpm_xlen, u16_t __angle) {
	uint_t static *xfs = nullptr, *yfs = nullptr, *pb_xlen = nullptr, *pm_xlen = nullptr, *pm_xfs;
	types::pixmap_t static pixbuff = nullptr, pixmap = nullptr;
	u16_t static *angle = nullptr;
	bool static inited = false;

	hipError_t any_err = hipSuccess;
	uint_t pb_size = __pm_ylen * __pb_xlen * 4;
	uint_t pm_size = __pm_ylen * __rpm_xlen * 4;
	if (!inited) {
		if ((any_err = hipMalloc((void **)&xfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'xfs', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&yfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'yfs', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&pb_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'pb_xlen', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&pm_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'pm_xlen', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&pm_xfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'pm_xfs', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if ((any_err = hipMalloc((void **)&angle, sizeof(u16_t))) != hipSuccess) {
			fprintf(stderr, "draw_pixmap: cuda, failed to alloc memory for 'angle', errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		if (__xfs == 0) {
			if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "draw_pixmap: cuda, failed to copy memory to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}

		if (__yfs == 0) {
			if ((any_err = hipMemcpy(yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "draw_pixmap: cuda, failed to copy memory to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}

		if (__pm_xfs == 0) {
			if ((any_err = hipMemcpy(pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "draw_pixmap: cuda, failed to copy memory to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}

		if (__angle == 0) {
			if ((any_err = hipMemcpy(angle, &__angle, sizeof(u16_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "draw_pixmap: cuda, failed to copy memory to device, errno: %d\n", any_err);
				return FFLY_FAILURE;
			}
		}
		inited = true;
	}

	uint_t static _pb_size{};
	if (_pb_size != pb_size)
	{
		if (pixbuff != nullptr) hipFree(pixbuff);
		if ((any_err = hipMalloc((void **)&pixbuff, pb_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pb_size = pb_size;
	}

	uint_t static _pm_size{};
	if (_pm_size != pm_size)
	{
		if (pixmap != nullptr) hipFree(pixmap);
		if ((any_err = hipMalloc((void **)&pixmap, pm_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_size = pm_size;
	}

	uint_t static _xfs{};
	if (_xfs != __xfs) {
		if ((any_err = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_xfs = __xfs;
	}

	uint_t static _yfs{};
	if (_yfs != __yfs) {
		if ((any_err = hipMemcpy(yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_yfs = __yfs;
	}

	uint_t static _pb_xlen{};
	if (_pb_xlen != __pb_xlen) {
		if ((any_err = hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pb_xlen = __pb_xlen;
	}

	uint_t static _pm_xlen{};
	if (_pm_xlen != __rpm_xlen) {
		if ((any_err = hipMemcpy(pm_xlen, &__rpm_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_xlen = __rpm_xlen;
	}

	uint_t static _pm_xfs{};
	if (_pm_xfs != __pm_xfs) {
		if ((any_err = hipMemcpy(pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}
		_pm_xfs = __pm_xfs;
	}

	u16_t static _angle = 0;
	if (_angle != __angle) {
		if ((any_err = hipMemcpy(angle, &__angle, sizeof(u16_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
			return FFLY_FAILURE;
		}

		_angle = __angle;
	}

	uint_t pb_fs = __yfs >= (__pb_ylen - __pm_ylen)? ((__pb_ylen - __pm_ylen) * __pb_xlen) * 4 : (__yfs * __pb_xlen) * 4;
	if ((any_err = hipMemcpy(pixbuff, __pixbuff + pb_fs, pb_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	if ((any_err = hipMemcpy(pixmap, __pixmap, pm_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}

	cu_draw_pixmap<<<__pm_ylen, __pm_xlen>>>(xfs, yfs, pixbuff, pb_xlen, pm_xfs, pixmap, pm_xlen, angle);
	if ((any_err = hipMemcpy(__pixbuff + pb_fs, pixbuff, pb_size * sizeof(types::__pixmap_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_err);
		return FFLY_FAILURE;
	}
	return FFLY_SUCCESS;
}
