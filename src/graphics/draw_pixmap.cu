#include "hip/hip_runtime.h"
# include "draw_pixmap.hpp"
__global__ void cu_draw_pixmap(mdl::uint_t *__xfs, mdl::uint_t *__yfs, mdl::firefly::types::pixmap_t __pixbuff, mdl::uint_t *__pb_xlen, mdl::uint_t *__pm_xfs, mdl::firefly::types::pixmap_t __pixmap, mdl::uint_t *__pm_xlen, boost::uint16_t *__angle) {
	mdl::uint_t pb_point = ((threadIdx.x + (*__xfs)) + (blockIdx.x * (*__pb_xlen))) * 4;
	mdl::uint_t pm_point = ((threadIdx.x + (*__pm_xfs)) + (blockIdx.x * (*__pm_xlen))) * 4;

	mdl::firefly::types::byte_t alpha = __pixmap[pm_point + 3];
	mdl::firefly::types::byte_t inv_alpha = 255 - __pixmap[pm_point + 3];

	if (__pixmap[pm_point + 3] != 0x0) {
		mdl::firefly::types::byte_t new_r = (mdl::firefly::types::byte_t)((alpha * __pixmap[pm_point] + inv_alpha * __pixbuff[pb_point]) >> 8);
		mdl::firefly::types::byte_t new_g = (mdl::firefly::types::byte_t)((alpha * __pixmap[pm_point + 1] + inv_alpha * __pixbuff[pb_point + 1]) >> 8);
		mdl::firefly::types::byte_t new_b = (mdl::firefly::types::byte_t)((alpha * __pixmap[pm_point + 2] + inv_alpha * __pixbuff[pb_point + 2]) >> 8);

		__pixbuff[pb_point] = new_r;
		__pixbuff[pb_point + 1] = new_g;
		__pixbuff[pb_point + 2] = new_b;

		__pixbuff[pb_point + 3] = __pixmap[pm_point + 3];
	}
}

// NOTE: not thread safe
mdl::firefly::types::err_t mdl::firefly::graphics::gpu_draw_pixmap(uint_t __xfs, uint_t __yfs, types::pixmap_t __pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, uint_t __pm_xfs, types::pixmap_t __pixmap, uint_t __pm_xlen, uint_t __pm_ylen, uint_t __rpm_xlen, boost::uint16_t __angle) {
	static uint_t *xfs = nullptr, *yfs = nullptr, *pb_xlen = nullptr, *pm_xlen = nullptr, *pm_xfs;
	static types::pixmap_t pixbuff = nullptr, pixmap = nullptr;
	static boost::uint16_t *angle = nullptr;
	static bool inited = false;
	hipError_t any_error = hipSuccess;

	if (!inited) {
		if ((any_error = hipMalloc((void **)&xfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if ((any_error = hipMalloc((void **)&yfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if ((any_error = hipMalloc((void **)&pb_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if ((any_error = hipMalloc((void **)&pm_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if ((any_error = hipMalloc((void **)&pm_xfs, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if ((any_error = hipMalloc((void **)&angle, sizeof(boost::uint16_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		// if the any of thease are 0 then it will skip the memcpy so in the kernel we could end up having some junk memory
		// that might cause an error.
		if (__xfs == 0) {
			if ((any_error = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__yfs == 0) {
			if ((any_error = hipMemcpy(yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__pb_xlen == 0) {
			if ((any_error = hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__rpm_xlen == 0) {
			if ((any_error = hipMemcpy(pm_xlen, &__rpm_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__pm_xfs == 0) {
			if ((any_error = hipMemcpy(pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__angle == 0) {
			if ((any_error = hipMemcpy(angle, &__angle, sizeof(boost::uint16_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}
		inited = true;
	}

	uint_t pixbuff_size = (__pm_ylen * __pb_xlen) * 4;
	uint_t pixmap_size = (__pm_ylen * __rpm_xlen) * 4;

	static uint_t _pixbuff_size = 0;
	if (_pixbuff_size != pixbuff_size) {
		if (pixbuff != nullptr) hipFree(pixbuff);

		if ((any_error = hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_pixbuff_size = pixbuff_size;
	}

	static uint_t _pixmap_size = 0;
	if (_pixmap_size != pixmap_size) {
		if (pixmap != nullptr) hipFree(pixmap);

		if ((any_error = hipMalloc((void **)&pixmap, pixmap_size * sizeof(types::__pixmap_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_pixmap_size = pixmap_size;
	}

	static uint_t _xfs = 0;
	if (_xfs != __xfs) {
		if ((any_error = hipMemcpy(xfs, &__xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_xfs = __xfs;
	}

	static uint_t _yfs = 0;
	if (_yfs != __yfs) {
		if ((any_error = hipMemcpy(yfs, &__yfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_yfs = __yfs;
	}

	static uint_t _pb_xlen = 0;
	if (_pb_xlen != __pb_xlen) {
		if ((any_error = hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_pb_xlen = __pb_xlen;
	}

	static uint_t _pm_xlen = 0;
	if (_pm_xlen != __rpm_xlen) {
		if ((any_error = hipMemcpy(pm_xlen, &__rpm_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_pm_xlen = __rpm_xlen;
	}

	static uint_t _pm_xfs = 0;
	if (_pm_xfs != __pm_xfs) {
		if ((any_error = hipMemcpy(pm_xfs, &__pm_xfs, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}
		_pm_xfs = __pm_xfs;
	}

	static boost::uint16_t _angle = 0;
	if (_angle != __angle) {
		if ((any_error = hipMemcpy(angle, &__angle, sizeof(boost::uint16_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_angle = __angle;
	}

	uint_t pb_offset = __yfs >= (__pb_ylen - __pm_ylen)? ((__pb_ylen - __pm_ylen) * __pb_xlen) * 4: (__yfs * __pb_xlen) * 4;
	if ((any_error = hipMemcpy(pixbuff, __pixbuff + pb_offset, pixbuff_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
		return FFLY_FAILURE;
	}

	if ((any_error = hipMemcpy(pixmap, __pixmap, pixmap_size * sizeof(types::__pixmap_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
		return FFLY_FAILURE;
	}

	cu_draw_pixmap<<<__pm_ylen, __pm_xlen>>>(xfs, yfs, pixbuff, pb_xlen, pm_xfs, pixmap, pm_xlen, angle);
	if ((any_error = hipMemcpy(__pixbuff + pb_offset, pixbuff, pixbuff_size * sizeof(types::__pixmap_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, errno: %d\n", any_error);
		return FFLY_FAILURE;
	}

	return FFLY_SUCCESS;
}
