#include "hip/hip_runtime.h"
# include "draw_pixmap.hpp"
__global__ void cu_draw_pixmap(mdl::uint_t *__xoffset, mdl::uint_t *__yoffset, boost::uint8_t *__pixbuff, mdl::uint_t *__pb_xlen, boost::uint8_t *__pixmap) {
	mdl::uint_t pixbuff_pos = ((threadIdx.x + (*__xoffset)) + ((blockIdx.x + (*__yoffset)) * (*__pb_xlen))) * 4;
	mdl::uint_t pixmap_pos = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;

	boost::uint8_t alpha = __pixmap[pixmap_pos + 3] + 1;
	boost::uint8_t inv_alpha = 256 - __pixmap[pixmap_pos + 3];
 
	if (__pixmap[pixmap_pos + 3] == 0) return;

	boost::uint8_t new_r = (boost::uint8_t)((alpha * __pixmap[pixmap_pos] + inv_alpha * __pixbuff[pixbuff_pos]) >> 8);
	boost::uint8_t new_g = (boost::uint8_t)((alpha * __pixmap[pixmap_pos + 1] + inv_alpha * __pixbuff[pixbuff_pos + 1]) >> 8);
	boost::uint8_t new_b = (boost::uint8_t)((alpha * __pixmap[pixmap_pos + 2] + inv_alpha * __pixbuff[pixbuff_pos + 2]) >> 8);

	__pixbuff[pixbuff_pos] = new_r;
	__pixbuff[pixbuff_pos + 1] = new_g;
	__pixbuff[pixbuff_pos + 2] = new_b;

	__pixbuff[pixbuff_pos + 3] = __pixmap[pixmap_pos + 3];
}

void mdl::firefly::graphics::draw_pixmap(uint_t __xoffset, uint_t __yoffset, boost::uint8_t *__pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, boost::uint8_t *__pixmap, uint_t __pm_xlen, uint_t __pm_ylen) {
	uint_t *xoffset, *yoffset, *pb_xlen;
	boost::uint8_t *pixbuff, *pixmap;

	hipMalloc((void **)&xoffset, sizeof(uint_t));
	hipMalloc((void **)&yoffset, sizeof(uint_t));
	hipMalloc((void **)&pb_xlen, sizeof(uint_t));

	uint_t pixbuff_size = (__pb_xlen * __pb_ylen) * 4;
	uint_t	pixmap_size = (__pm_xlen * __pm_ylen) * 4;

	hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(boost::uint8_t));
	hipMalloc((void **)&pixmap, pixmap_size * sizeof(boost::uint8_t));

	hipMemcpy(xoffset, &__xoffset, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(yoffset, &__xoffset, sizeof(uint_t), hipMemcpyHostToDevice);
	hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice);

	hipMemcpy(pixbuff, __pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(pixmap, __pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice);

	cu_draw_pixmap<<<__pm_ylen, __pm_xlen>>>(xoffset, yoffset, pixbuff, pb_xlen, pixmap);

	hipMemcpy(__pixbuff, pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost);

	hipFree(xoffset);
	hipFree(yoffset);
	hipFree(pb_xlen);
	hipFree(pixbuff);
	hipFree(pixmap);
}
