#include "hip/hip_runtime.h"
# include "draw_pixmap.hpp"
__global__ void cu_draw_pixmap(mdl::uint_t *__xoffset, mdl::uint_t *__yoffset, boost::uint8_t *__pixbuff, mdl::uint_t *__pb_xlen, boost::uint8_t *__pixmap) {
	mdl::uint_t pixbuff_pos = ((threadIdx.x + (*__xoffset)) + ((blockIdx.x + (*__yoffset)) * (*__pb_xlen))) * 4;
	mdl::uint_t pixmap_pos = (threadIdx.x + (blockIdx.x * blockDim.x)) * 4;

	boost::uint8_t alpha = __pixmap[pixmap_pos + 3];
	boost::uint8_t inv_alpha = 255 - __pixmap[pixmap_pos + 3];
 
	if (__pixmap[pixmap_pos + 3] != 0x0) {

	boost::uint8_t new_r = (boost::uint8_t)((alpha * __pixmap[pixmap_pos] + inv_alpha * __pixbuff[pixbuff_pos]) >> 8);
	boost::uint8_t new_g = (boost::uint8_t)((alpha * __pixmap[pixmap_pos + 1] + inv_alpha * __pixbuff[pixbuff_pos + 1]) >> 8);
	boost::uint8_t new_b = (boost::uint8_t)((alpha * __pixmap[pixmap_pos + 2] + inv_alpha * __pixbuff[pixbuff_pos + 2]) >> 8);

	__pixbuff[pixbuff_pos] = new_r;
	__pixbuff[pixbuff_pos + 1] = new_g;
	__pixbuff[pixbuff_pos + 2] = new_b;

	__pixbuff[pixbuff_pos + 3] = __pixmap[pixmap_pos + 3];

	}
}

// NOTE: not thread safe
boost::int8_t mdl::firefly::graphics::draw_pixmap(uint_t __xoffset, uint_t __yoffset, boost::uint8_t *__pixbuff, uint_t __pb_xlen, uint_t __pb_ylen, boost::uint8_t *__pixmap, uint_t __pm_xlen, uint_t __pm_ylen) {
	static uint_t *xoffset = nullptr, *yoffset = nullptr, *pb_xlen = nullptr;
	static boost::uint8_t *pixbuff = nullptr, *pixmap = nullptr;
	static bool initialized = false;
	hipError_t any_error = hipSuccess;

	if (!initialized) {
		if ((any_error = hipMalloc((void **)&xoffset, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}
	
		if ((any_error = hipMalloc((void **)&yoffset, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		if ((any_error = hipMalloc((void **)&pb_xlen, sizeof(uint_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		// if the any of thease are 0 then it will skip the memcpy so in the kernel we could end up having some junk memory
		// that might cause an error.
		if (__xoffset == 0) {
			if ((any_error = hipMemcpy(xoffset, &__xoffset, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__yoffset == 0) {
			if ((any_error = hipMemcpy(yoffset, &__yoffset, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		if (__pm_xlen == 0) {
			if ((any_error = hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
				return FFLY_FAILURE;
			}
		}

		initialized = true;
	}

	uint_t pixbuff_size = (__pb_xlen * __pb_ylen) * 4;
	uint_t pixmap_size = (__pm_xlen * __pm_ylen) * 4;

	static uint_t _pixbuff_size = 0;

	if (_pixbuff_size != pixbuff_size) {
		if (pixbuff != nullptr) hipFree(pixbuff);

		if ((any_error = hipMalloc((void **)&pixbuff, pixbuff_size * sizeof(boost::uint8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_pixbuff_size = pixbuff_size;
	}

	static uint_t _pixmap_size = 0;

	if (_pixmap_size != pixmap_size) {
		if (pixmap != nullptr) hipFree(pixmap);

		if ((any_error = hipMalloc((void **)&pixmap, pixmap_size * sizeof(boost::uint8_t))) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Malloc, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_pixmap_size = pixmap_size;
	}

	static uint_t _xoffset = 0;
	
	if (_xoffset != __xoffset) {
		if ((any_error = hipMemcpy(xoffset, &__xoffset, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_xoffset = __xoffset;
	}


	static uint_t _yoffset = 0;

	if (_yoffset != __yoffset) {
		if ((any_error = hipMemcpy(yoffset, &__yoffset, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_yoffset = __yoffset;
	}


	static uint_t _pb_xlen = 0;

	if (_pb_xlen != __pb_xlen) { 
		if ((any_error = hipMemcpy(pb_xlen, &__pb_xlen, sizeof(uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
			fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
			return FFLY_FAILURE;
		}

		_pb_xlen = __pb_xlen;
	}

	if ((any_error = hipMemcpy(pixbuff, __pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		return FFLY_FAILURE;
	}

	if ((any_error = hipMemcpy(pixmap, __pixmap, pixmap_size * sizeof(boost::uint8_t), hipMemcpyHostToDevice)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		return FFLY_FAILURE;
	}

	cu_draw_pixmap<<<__pm_ylen, __pm_xlen>>>(xoffset, yoffset, pixbuff, pb_xlen, pixmap);

	if ((any_error = hipMemcpy(__pixbuff, pixbuff, pixbuff_size * sizeof(boost::uint8_t), hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "cuda: failed to call Memcpy, error code: %d\n", any_error);
		return FFLY_FAILURE;
	}

	return FFLY_SUCCESS;
}
