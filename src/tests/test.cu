#include "hip/hip_runtime.h"
# include <cstdint>
# include <cstdio>
__global__ void test(uint8_t *__ptr, size_t __pitch) {
	int x = threadIdx.x;
	uint8_t *row = (uint8_t*)((char *)__ptr + x * __pitch);
	row[13] = 'X';
/*
	for (int o = 0; o != 10240; o ++) {
		if (*(__ptr + o) == 'K') {
			printf("%d offset\n", o);
			break;
		}
	}
*/
}

# include <cstdlib>
# include <iostream>
int main() {
	int unsigned xlen = 20, ylen = 20;

	uint8_t *large_data = (uint8_t*)malloc(xlen * ylen * sizeof(uint8_t));

	uint8_t **host_data = (uint8_t **)malloc(ylen * sizeof(uint8_t *));
	//uint8_t large_data[ylen * xlen + 200];
//	uint8_t host_data[ylen][xlen];
	for (int unsigned y{}; y != ylen; y ++) {
		host_data[y] = &large_data[y * xlen];
		for (int unsigned x{}; x != xlen; x ++) {
			host_data[y][x] = ' ';
		}
	}

	uint8_t *device_data = nullptr;

	size_t pitch;

	hipMallocPitch((void **)&device_data, &pitch, xlen * sizeof(uint8_t) - 2, ylen);

	printf("pitch: %d\n", pitch);
	if (hipMemcpy2D(device_data, pitch, host_data[0], xlen * sizeof(uint8_t), xlen * sizeof(uint8_t) - 2, ylen, hipMemcpyHostToDevice) != hipSuccess){
		fprintf(stderr, "failed to copy memory to device.\n");
		return 1;
	}

	test<<<1, xlen>>>(device_data, pitch);
	hipDeviceSynchronize();

	if (hipMemcpy2D(host_data[0], xlen * sizeof(uint8_t), device_data, pitch, xlen * sizeof(uint8_t) - 2, ylen, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "failed to copy memory to host.\n");
		return 1;
	}

	for (int unsigned y{}; y != ylen; y ++) {
		for (int unsigned x{}; x != xlen; x ++)
			printf("%c", host_data[y][x]);
		printf("\n");
	}

	hipDeviceReset();

	std::free(large_data);
	std::free(host_data);
	hipFree(device_data);
}

