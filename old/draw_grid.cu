#include "hip/hip_runtime.h"
# include "draw_grid.hpp"
# define GRID_THIKNESS 8
# define GRID_CR 63
# define GRID_CG 65
# define GRID_CB 89
# define GRID_CA 255
__global__ void _draw_grid(boost::uint8_t *__pixmap, mdl::uint_t *__offsets, mdl::uint_t *xlen) {

	for (std::size_t o = (threadIdx.x * __offsets[0]); o != GRID_THIKNESS + (threadIdx.x * __offsets[0]); o ++) {
		mdl::uint_t pix_pos = (o + ((blockIdx.x * __offsets[1]) * (*xlen))) * 4;
		__pixmap[pix_pos] = GRID_CR;
		__pixmap[pix_pos + 1] = GRID_CG;
		__pixmap[pix_pos + 2] = GRID_CB;
		__pixmap[pix_pos + 3] = GRID_CA;
	}
}

# include <cstdio>
boost::uint8_t *pixmap;
mdl::uint_t *offsets, *xlen;
void draw_grid(boost::uint8_t *__pixmap, mdl::uint_t __size[2], mdl::uint_t __offsets[2]) {
	mdl::uint_t xx = GRID_THIKNESS * (__size[0] / __offsets[0]);
	mdl::uint_t yy = (__size[1] / __offsets[1]);

	mdl::uint_t size = (__size[0] * __size[1]) * 4;
	hipMalloc((void **)&pixmap, size);
	hipMalloc((void **)&offsets, 2 * sizeof(mdl::uint_t));
	hipMalloc((void **)&xlen, sizeof(mdl::uint_t));

	hipMemcpy(pixmap, __pixmap, size, hipMemcpyHostToDevice);

	hipMemcpy(offsets, __offsets, 2 * sizeof(mdl::uint_t), hipMemcpyHostToDevice);

	hipMemcpy(xlen, &__size[0], sizeof(mdl::uint_t), hipMemcpyHostToDevice);

	_draw_grid<<<yy, xx>>>(pixmap, offsets, xlen);

	hipMemcpy(__pixmap, pixmap, size, hipMemcpyDeviceToHost);

	hipFree(pixmap);
	hipFree(offsets);
}
