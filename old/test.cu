
#include <hip/hip_runtime.h>
# include <cstdint>
# include <cstdio>
__global__ void cu_test(uint8_t *__data) {
	__data[threadIdx.x] = threadIdx.x;
}

uint8_t *test() {
	uint8_t *data = (uint8_t *)malloc(8);
	uint8_t *_data;
	hipError_t any_error = hipSuccess;

	if ((any_error = hipMalloc((void **)&_data, 8)) != hipSuccess) {
		fprintf(stderr, "Malloc failed with error code: %d\n", any_error);
		return nullptr;
	}

	cu_test<<<1, 8>>>(_data);

	if ((any_error = hipMemcpy(data, _data, 8, hipMemcpyDeviceToHost)) != hipSuccess) {
		fprintf(stderr, "Memcpy failed with error code: %d\n", any_error);
		return nullptr;
	}

	hipFree(_data);

	return data;
}
